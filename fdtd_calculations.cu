#include "hip/hip_runtime.h"
#include "fdtd_calculations.h"


#define OFFSET(p, x, y, z) (p[(z)*ny*nx + (y)*nx + (x)])


__global__ void updateHField(float *hx,       float *hy,       float *hz,
                             float *exSource, float *eySource, float *ezSource,
                             int nx, int ny, int nz,
                             float dt, float dx, float dy, float dz,
                             float mu0)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int iz = threadIdx.z + blockIdx.z * blockDim.z;

    // Update hx
    if(ix > 0 && ix < nx-1 &&
       iy > 0 && iy < ny-1 &&
       iz > 0 && iz < nz-1) {
        OFFSET(hx, ix, iy, iz) = OFFSET(hx, ix, iy, iz) -
                                 dt/(mu0 * dy) *
                                 (OFFSET(ezSource, ix, iy+1, iz) - OFFSET(ezSource, ix, iy, iz)) +
                                 dt/(mu0 * dz) *
                                 (OFFSET(eySource, ix, iy, iz+1) - OFFSET(eySource, ix, iy, iz));
    }
    
    // Update hy
    if(ix > 0 && ix < nx-1 &&
       iy > 1 && iy < ny-1 &&
       iz > 0 && iz < nz-1) {
        OFFSET(hy, ix, iy, iz) = OFFSET(hy, ix, iy, iz) -
                                 dt/(mu0 * dz) *
                                 (OFFSET(exSource, ix, iy, iz+1) - OFFSET(exSource, ix, iy, iz)) +
                                 dt/(mu0 * dx) *
                                 (OFFSET(ezSource, ix+1, iy, iz) - OFFSET(ezSource, ix, iy, iz));
    }
    
    // Update hz
    if(ix > 0 && ix < nx-1 &&
       iy > 0 && iy < ny-1 &&
       iz > 1 && iz < nz-1) {
        OFFSET(hz, ix, iy, iz) = OFFSET(hz, ix, iy, iz) -
                                 dt/(mu0 * dx) *
                                 (OFFSET(eySource, ix+1, iy, iz) - OFFSET(eySource, ix, iy, iz)) +
                                 dt/(mu0 * dy) *
                                 (OFFSET(exSource, ix, iy+1, iz) - OFFSET(exSource, ix, iy, iz));
    }
}


__global__ void updateDField(float *dxTarget, float *dyTarget, float *dzTarget,
                             float *dxSource, float *dySource, float *dzSource,
                             float *hx,       float *hy,       float *hz,
                             int nx, int ny, int nz,
                             float dt, float dx, float dy, float dz)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int iz = threadIdx.z + blockIdx.z * blockDim.z;

    // Update dx
    if(ix > 0 && ix < nx-1 &&
       iy > 1 && iy < ny-1 &&
       iz > 1 && iz < nz-1) {
        OFFSET(dxSource, ix, iy, iz) = OFFSET(dxSource, ix, iy, iz) +
                                       dt/dy * (OFFSET(hz, ix, iy, iz) - OFFSET(hz, ix, iy-1, iz)) -
                                       dt/dz * (OFFSET(hy, ix, iy, iz) - OFFSET(hy, ix, iy, iz-1));
    }
    
    // Update dy
    if(ix > 1 && ix < nx-1 &&
       iy > 0 && iy < ny-1 &&
       iz > 1 && iz < nz-1) {
        OFFSET(dyTarget, ix, iy, iz) = OFFSET(dySource, ix, iy, iz) +
                                       dt/dz * (OFFSET(hx, ix, iy, iz) - OFFSET(hx, ix, iy, iz-1)) -
                                       dt/dx * (OFFSET(hz, ix, iy, iz) - OFFSET(hz, ix-1, iy, iz));
    }
    
    // Update dz
    if(ix > 1 && ix < nx-1 &&
       iy > 1 && iy < ny-1 &&
       iz > 0 && iz < nz-1) {
            OFFSET(dzTarget, ix, iy, iz) = OFFSET(dzSource, ix, iy, iz) +
                                           dt/dx * (OFFSET(hy, ix, iy, iz) - OFFSET(hy, ix-1, iy, iz)) -
                                           dt/dy * (OFFSET(hx, ix, iy, iz) - OFFSET(hx, ix, iy-1, iz));
    }
}


__global__ void updateEField(float *exTarget,  float *eyTarget,  float *ezTarget,
                             float *exSource0, float *eySource0, float *ezSource0,
                             float *exSource1, float *eySource1, float *ezSource1,
                             float *dxSource0, float *dySource0, float *dzSource0,
                             float *dxSource1, float *dySource1, float *dzSource1,
                             float *dxSource2, float *dySource2, float *dzSource2,
                             float *sigma,float *epsI, float *epsS, float *tauD,
                             int nx, int ny, int nz,
                             float dt, float eps0)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int iz = threadIdx.z + blockIdx.z * blockDim.z;

    // Update ex
    if(ix > 0 && ix < nx-1 &&
       iy > 1 && iy < ny-1 &&
       iz > 1 && iz < nz-1) {
        OFFSET(exTarget, ix, iy, iz) = (
                                        1.0/(2.0 * eps0 * OFFSET(epsI, ix, iy, iz) * OFFSET(tauD, ix, iy, iz) +
                                        2.0 * dt *
                                        (
                                         eps0 * OFFSET(epsS, ix, iy, iz) +
                                         OFFSET(sigma, ix, iy, iz) * OFFSET(tauD, ix, iy, iz)
                                        ) +
                                        OFFSET(sigma, ix, iy, iz) * dt * dt)
                                       ) *
                                       (
                                        (
                                         4.0 * eps0 * OFFSET(epsI, ix, iy, iz) * OFFSET(tauD, ix, iy, iz) +
                                         2.0 * dt *
                                         (
                                          eps0 * OFFSET(epsS, ix, iy, iz) +
                                          OFFSET(sigma, ix, iy, iz) * OFFSET(tauD, ix, iy, iz)
                                         ) -
                                         OFFSET(sigma, ix, iy, iz) * dt * dt
                                        ) *
                                        OFFSET(exSource0, ix, iy, iz) -
                                        (2.0 * eps0 * OFFSET(epsI, ix, iy, iz) * OFFSET(tauD, ix, iy, iz)) *
                                        OFFSET(exSource1, ix, iy, iz) +
                                        (2.0 * (dt + OFFSET(tauD, ix, iy, iz))) * OFFSET(dxSource0, ix, iy, iz) -
                                        (2.0 * dt + 4.0 * OFFSET(tauD, ix, iy, iz)) * OFFSET(dxSource1, ix, iy, iz) +
                                        (2.0 * OFFSET(tauD, ix, iy, iz)) * OFFSET(dxSource2, ix, iy, iz)
                                       );
    }
    
    // Update ey
    if(ix >= 2 && ix <= nx-1 &&
       iy >= 1 && iy <= ny-1 &&
       iz >= 2 && iz <= nz-1) {
        OFFSET(eyTarget, ix, iy, iz) = (
                                        1.0/(2.0 * eps0 * OFFSET(epsI, ix, iy, iz) * OFFSET(tauD, ix, iy, iz) +
                                        2.0 * dt *
                                        (
                                         eps0 * OFFSET(epsS, ix, iy, iz) +
                                         OFFSET(sigma, ix, iy, iz) * OFFSET(tauD, ix, iy, iz)
                                        ) +
                                        OFFSET(sigma, ix, iy, iz) * dt * dt)
                                       ) *
                                       (
                                        (
                                         4.0 * eps0 * OFFSET(epsI, ix, iy, iz) * OFFSET(tauD, ix, iy, iz) +
                                         2.0 * dt *
                                         (
                                          eps0 * OFFSET(epsS, ix, iy, iz) +
                                          OFFSET(sigma, ix, iy, iz) * OFFSET(tauD, ix, iy, iz)
                                         ) -
                                         OFFSET(sigma, ix, iy, iz) * dt * dt
                                        ) *
                                        OFFSET(eySource0, ix, iy, iz) -
                                        (2.0 * eps0 * OFFSET(epsI, ix, iy, iz) * OFFSET(tauD, ix, iy, iz)) *
                                        OFFSET(eySource1, ix, iy, iz) +
                                        (2.0 * (dt + OFFSET(tauD, ix, iy, iz))) * OFFSET(dySource0, ix, iy, iz) -
                                        (2.0 * dt + 4.0 * OFFSET(tauD, ix, iy, iz)) * OFFSET(dySource1, ix, iy, iz) +
                                        (2.0 * OFFSET(tauD, ix, iy, iz)) * OFFSET(dySource2, ix, iy, iz)
                                       );
    }
    
    // Update ez
    if(ix >= 2 && ix <= nx-1 &&
       iy >= 1 && iy <= ny-1 &&
       iz >= 2 && iz <= nz-1) {
        OFFSET(ezTarget, ix, iy, iz) = (
                                        1.0/(2.0 * eps0 * OFFSET(epsI, ix, iy, iz) * OFFSET(tauD, ix, iy, iz) +
                                        2.0 * dt *
                                        (
                                         eps0 * OFFSET(epsS, ix, iy, iz) +
                                         OFFSET(sigma, ix, iy, iz) * OFFSET(tauD, ix, iy, iz)
                                        ) +
                                        OFFSET(sigma, ix, iy, iz) * dt * dt)
                                       ) *
                                       (
                                        (
                                         4.0 * eps0 * OFFSET(epsI, ix, iy, iz) * OFFSET(tauD, ix, iy, iz) +
                                         2.0 * dt *
                                         (
                                          eps0 * OFFSET(epsS, ix, iy, iz) +
                                          OFFSET(sigma, ix, iy, iz) * OFFSET(tauD, ix, iy, iz)
                                         ) -
                                         OFFSET(sigma, ix, iy, iz) * dt * dt
                                        ) *
                                        OFFSET(ezSource0, ix, iy, iz) -
                                        (2.0 * eps0 * OFFSET(epsI, ix, iy, iz) * OFFSET(tauD, ix, iy, iz)) *
                                        OFFSET(ezSource1, ix, iy, iz) +
                                        (2.0 * (dt + OFFSET(tauD, ix, iy, iz))) * OFFSET(dzSource0, ix, iy, iz) -
                                        (2.0 * dt + 4.0 * OFFSET(tauD, ix, iy, iz)) * OFFSET(dzSource1, ix, iy, iz) +
                                        (2.0 * OFFSET(tauD, ix, iy, iz)) * OFFSET(dzSource2, ix, iy, iz)
                                       );
    }
}


__global__ void updateSource(float *dzTarget, float *dzSource,
                             float *hx,       float *hy,
                             int *src, float *jz,
                             float dt, float dx, float dy, float dz,
                             int nsrc, int runsCount)
{
}


__global__ void updateMurBoundary(float *exTarget, float *eyTarget, float *ezTarget,
                                  float *exSource, float *eySource, float *ezSource,
                                  float *rpx0,     float *rpy0,     float *rpz0,
                                  float *rpxEnd,   float *rpyEnd,   float *rpZend,
                                  int nx, int ny, int nz,
                                  float dt, float dx, float dy, float dz,
                                  float mu0, float eps0)
{
}
