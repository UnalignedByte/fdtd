#include "hip/hip_runtime.h"
#include "fdtd.h"

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>

#include "utils.h"
#include "fdtd_calculations.h"


#define BLOCK_X 128
#define BLOCK_Y 1
#define BLOCK_Z 1


int main(int argc, char **argv)
{
    // Read params
    FdtdParams *params;
    printf("Reading parameters...\n");
    params = initParamsWithPath("data/input_params");
    printParams(params);

    // Initialize field
    FdtdField  *field, *deviceField; // Used for CUDA

    printf("Initializing field...\n");
    field = initFieldWithParams(params);
    setupMurBoundary(params, field);

    printf("Initializing device field...\n");
    deviceField = initDeviceFieldWithParams(params);

    printf("Reading materials data...\n");
    loadMaterials(params, field, "data/mat_specs_riken", params->inputPath);

    printf("Initializing sources...\n");
    setupSources(params);

    printf("Copying data to GPU...\n\n");
    copyDataToDevice(params, field, deviceField);
    copySymbolsToDevice(params);

    // Setup CUDA parameters
    dim3 gridSize = dim3((params->nx + BLOCK_X - 1)/BLOCK_X,
                         (params->ny + BLOCK_Y - 1)/BLOCK_Y,
                         (params->nz + BLOCK_Z - 1)/BLOCK_Z);
    dim3 blockSize = dim3(BLOCK_X, BLOCK_Y, BLOCK_Z);

    // Create streams
    hipStream_t streamH;
    hipStream_t streamD;
    hipStream_t streamE;

    CHECK(hipStreamCreate(&streamH))
    CHECK(hipStreamCreate(&streamD))
    CHECK(hipStreamCreate(&streamE))
    
    hipEvent_t eventH;
    hipEvent_t eventD;
    hipEvent_t eventE;

    CHECK(hipEventCreate(&eventH))
    CHECK(hipEventCreate(&eventD))
    CHECK(hipEventCreate(&eventE))
    
    int bytesCount = params->nx * params->ny * params->nz * sizeof(float); 

    // Threads
    CopyParams *hCopyParams;
    CopyParams *dCopyParams;
    CopyParams *eCopyParams;

    pthread_t *hThread = NULL;
    pthread_t *dThread = NULL;
    pthread_t *eThread = NULL;

    ResultsParams *resultsParams;
    pthread_t *threads = (pthread_t *)malloc(params->iterationsCount * sizeof(pthread_t));

    CHECK(hipEventRecord(eventE))

    // Main loop
    for(int i=0; i<params->iterationsCount; i += 3) {

        // Run 0
        printf("Running iteration %d\n", i);

        // H field
        CHECK(hipStreamWaitEvent(streamH, eventE, 0));

        if(hThread != NULL)
            pthread_join(*hThread, NULL);

        updateHField<<<gridSize, blockSize, 0, streamH>>>(deviceField->hx,  deviceField->hy,  deviceField->hz,                    
                                                          deviceField->ex2, deviceField->ey2, deviceField->ez2);

        CHECK(hipEventRecord(eventH, streamH));

        CHECK(hipMemcpyAsync(field->hx, deviceField->hx, bytesCount, hipMemcpyHostToDevice, streamH));
        CHECK(hipMemcpyAsync(field->hy, deviceField->hy, bytesCount, hipMemcpyHostToDevice, streamH));
        CHECK(hipMemcpyAsync(field->hz, deviceField->hz, bytesCount, hipMemcpyHostToDevice, streamH));

        // Spawn copy thread
        hCopyParams = (CopyParams *)malloc(sizeof(CopyParams));
        hCopyParams->xSource = field->hx;
        hCopyParams->ySource = field->hy;
        hCopyParams->zSource = field->hz;
        hCopyParams->params = params;
        hCopyParams->stream = streamH;

        hThread = (pthread_t *)malloc(sizeof(pthread_t));
        pthread_create(hThread, NULL, copyResultsWithParams, hCopyParams);

        // D field
        CHECK(hipStreamWaitEvent(streamD, eventH, 0));

        if(dThread != NULL)
            pthread_join(*dThread, NULL);

        updateDField<<<gridSize, blockSize, 0, streamD>>>(deviceField->dx0, deviceField->dy0, deviceField->dz0, 
                                                          deviceField->dx2, deviceField->dy2, deviceField->dz2, 
                                                          deviceField->hx,  deviceField->hy,  deviceField->hz);
 
        updateSources<<<gridSize, blockSize, 0, streamD>>>(deviceField->dz0, deviceField->dz2,
                                                           deviceField->hx,  deviceField->hy,
                                                           i);

        CHECK(hipEventRecord(eventD, streamD));

        CHECK(hipMemcpyAsync(field->dx0, deviceField->dx0, bytesCount, hipMemcpyDeviceToHost, streamD))
        CHECK(hipMemcpyAsync(field->dy0, deviceField->dy0, bytesCount, hipMemcpyDeviceToHost, streamD))
        CHECK(hipMemcpyAsync(field->dz0, deviceField->dz0, bytesCount, hipMemcpyDeviceToHost, streamD))

        // Spawn copy thread
        dCopyParams = (CopyParams *)malloc(sizeof(CopyParams));
        dCopyParams->xSource = field->dx0;
        dCopyParams->ySource = field->dy0;
        dCopyParams->zSource = field->dz0;
        dCopyParams->params = params;
        dCopyParams->stream = streamD;

        dThread = (pthread_t *)malloc(sizeof(pthread_t));
        pthread_create(dThread, NULL, copyResultsWithParams, dCopyParams);

        // E field
        CHECK(hipStreamWaitEvent(streamE, eventD, 0));

        if(eThread != NULL)
            pthread_join(*eThread, NULL);

        updateEField<<<gridSize, blockSize, 0, streamE>>>(deviceField->ex0, deviceField->ey0, deviceField->ez0, 
                                                          deviceField->ex2, deviceField->ey2, deviceField->ez2, 
                                                          deviceField->ex1, deviceField->ey1, deviceField->ez1, 
                                                          deviceField->dx0, deviceField->dy0, deviceField->dz0, 
                                                          deviceField->dx2, deviceField->dy2, deviceField->dz2, 
                                                          deviceField->dx1, deviceField->dy1, deviceField->dz1, 
                                                          deviceField->sigma, deviceField->epsI, deviceField->epsS, deviceField->tauD);
            
        updateMurBoundary<<<gridSize, blockSize, 0, streamE>>>(deviceField->ex0,  deviceField->ey0,  deviceField->ez0,                 
                                                               deviceField->ex2,  deviceField->ey2,  deviceField->ez2,                 
                                                               deviceField->rpx0, deviceField->rpy0, deviceField->rpz0,                         
                                                               deviceField->rpxEnd, deviceField->rpyEnd, deviceField->rpzEnd);

        CHECK(hipEventRecord(eventE, streamE));

        CHECK(hipMemcpyAsync(field->ex0, deviceField->ex0, bytesCount, hipMemcpyDeviceToHost, streamE))
        CHECK(hipMemcpyAsync(field->ey0, deviceField->ey0, bytesCount, hipMemcpyDeviceToHost, streamE))
        CHECK(hipMemcpyAsync(field->ez0, deviceField->ez0, bytesCount, hipMemcpyDeviceToHost, streamE))

        // Spawn copy thread
        eCopyParams = (CopyParams *)malloc(sizeof(CopyParams));
        eCopyParams->xSource = field->ex0;
        eCopyParams->ySource = field->ey0;
        eCopyParams->zSource = field->ez0;
        eCopyParams->params = params;
        eCopyParams->stream = streamD;

        eThread = (pthread_t *)malloc(sizeof(pthread_t));
        pthread_create(eThread, NULL, copyResultsWithParams, eCopyParams);

        //Spawn write results thread
        resultsParams = (ResultsParams *)malloc(sizeof(ResultsParams));
        resultsParams->params = params;
        resultsParams->field = field;
        resultsParams->hParams = hCopyParams;
        resultsParams->dParams = dCopyParams;
        resultsParams->eParams = eCopyParams;
        resultsParams->hThread = hThread;
        resultsParams->dThread = dThread;
        resultsParams->eThread = eThread;
        resultsParams->currentIteration = i;

        pthread_create(&threads[i], NULL, writeResultsWithParams, resultsParams);

        // Run 1
        printf("Running iteration %d\n", i+1);

        // H field
        CHECK(hipStreamWaitEvent(streamH, eventE, 0));

        if(hThread != NULL)
            pthread_join(*hThread, NULL);

        updateHField<<<gridSize, blockSize, 0, streamH>>>(deviceField->hx,  deviceField->hy,  deviceField->hz,                    
                                                          deviceField->ex0, deviceField->ey0, deviceField->ez0);

        CHECK(hipEventRecord(eventH, streamH));
    
        CHECK(hipMemcpyAsync(field->hx, deviceField->hx, bytesCount, hipMemcpyHostToDevice, streamH));
        CHECK(hipMemcpyAsync(field->hy, deviceField->hy, bytesCount, hipMemcpyHostToDevice, streamH));
        CHECK(hipMemcpyAsync(field->hz, deviceField->hz, bytesCount, hipMemcpyHostToDevice, streamH));

        // Spawn copy thread
        hCopyParams = (CopyParams *)malloc(sizeof(CopyParams));
        hCopyParams->xSource = field->hx;
        hCopyParams->ySource = field->hy;
        hCopyParams->zSource = field->hz;
        hCopyParams->params = params;
        hCopyParams->stream = streamH;

        hThread = (pthread_t *)malloc(sizeof(pthread_t));
        pthread_create(hThread, NULL, copyResultsWithParams, hCopyParams);

        // D field
        CHECK(hipStreamWaitEvent(streamD, eventH, 0));

        updateDField<<<gridSize, blockSize, 0, streamD>>>(deviceField->dx1, deviceField->dy1, deviceField->dz1, 
                                                          deviceField->dx0, deviceField->dy0, deviceField->dz0, 
                                                          deviceField->hx,  deviceField->hy,  deviceField->hz);
 
        updateSources<<<gridSize, blockSize, 0, streamD>>>(deviceField->dz1, deviceField->dz0,
                                                           deviceField->hx,  deviceField->hy,
                                                           i);

        CHECK(hipEventRecord(eventD, streamD));

        CHECK(hipMemcpyAsync(field->dx0, deviceField->dx1, bytesCount, hipMemcpyDeviceToHost, streamD))
        CHECK(hipMemcpyAsync(field->dy0, deviceField->dy1, bytesCount, hipMemcpyDeviceToHost, streamD))
        CHECK(hipMemcpyAsync(field->dz0, deviceField->dz1, bytesCount, hipMemcpyDeviceToHost, streamD))

        // Spawn copy thread
        dCopyParams = (CopyParams *)malloc(sizeof(CopyParams));
        dCopyParams->xSource = field->dx0;
        dCopyParams->ySource = field->dy0;
        dCopyParams->zSource = field->dz0;
        dCopyParams->params = params;
        dCopyParams->stream = streamD;

        dThread = (pthread_t *)malloc(sizeof(pthread_t));
        pthread_create(dThread, NULL, copyResultsWithParams, dCopyParams);
 
        // E field
        CHECK(hipStreamWaitEvent(streamE, eventD, 0));

        updateEField<<<gridSize, blockSize, 0, streamE>>>(deviceField->ex1, deviceField->ey1, deviceField->ez1,
                                                          deviceField->ex0, deviceField->ey0, deviceField->ez0,
                                                          deviceField->ex2, deviceField->ey2, deviceField->ez2,
                                                          deviceField->dx1, deviceField->dy1, deviceField->dz1,
                                                          deviceField->dx0, deviceField->dy0, deviceField->dz0,
                                                          deviceField->dx2, deviceField->dy2, deviceField->dz2,
                                                          deviceField->sigma, deviceField->epsI, deviceField->epsS, deviceField->tauD);
            
        updateMurBoundary<<<gridSize, blockSize, 0, streamE>>>(deviceField->ex1,  deviceField->ey1,  deviceField->ez1,                 
                                                               deviceField->ex0,  deviceField->ey0,  deviceField->ez0,                 
                                                               deviceField->rpx0, deviceField->rpy0, deviceField->rpz0,                         
                                                               deviceField->rpxEnd, deviceField->rpyEnd, deviceField->rpzEnd);

        CHECK(hipEventRecord(eventE, streamE));

        CHECK(hipMemcpyAsync(field->ex0, deviceField->ex1, bytesCount, hipMemcpyDeviceToHost, streamE))
        CHECK(hipMemcpyAsync(field->ey0, deviceField->ey1, bytesCount, hipMemcpyDeviceToHost, streamE))
        CHECK(hipMemcpyAsync(field->ez0, deviceField->ez1, bytesCount, hipMemcpyDeviceToHost, streamE))

        // Spawn copy thread
        eCopyParams = (CopyParams *)malloc(sizeof(CopyParams));
        eCopyParams->xSource = field->ex0;
        eCopyParams->ySource = field->ey0;
        eCopyParams->zSource = field->ez0;
        eCopyParams->params = params;
        eCopyParams->stream = streamD;

        eThread = (pthread_t *)malloc(sizeof(pthread_t));
        pthread_create(eThread, NULL, copyResultsWithParams, eCopyParams);

        //Spawn write results thread
        resultsParams = (ResultsParams *)malloc(sizeof(ResultsParams));
        resultsParams->params = params;
        resultsParams->field = field;
        resultsParams->hParams = hCopyParams;
        resultsParams->dParams = dCopyParams;
        resultsParams->eParams = eCopyParams;
        resultsParams->hThread = hThread;
        resultsParams->dThread = dThread;
        resultsParams->eThread = eThread;
        resultsParams->currentIteration = i+1;

        pthread_create(&threads[i+1], NULL, writeResultsWithParams, resultsParams);

        // Run 2
        printf("Running iteration %d\n", i+2);

        // H field
        CHECK(hipStreamWaitEvent(streamH, eventE, 0));

        if(hThread != NULL)
            pthread_join(*hThread, NULL);

        updateHField<<<gridSize, blockSize, 0, streamH>>>(deviceField->hx,  deviceField->hy,  deviceField->hz,                    
                                                          deviceField->ex1, deviceField->ey1, deviceField->ez1);

        CHECK(hipEventRecord(eventH, streamH));

        CHECK(hipMemcpyAsync(field->hx, deviceField->hx, bytesCount, hipMemcpyHostToDevice, streamH));
        CHECK(hipMemcpyAsync(field->hy, deviceField->hy, bytesCount, hipMemcpyHostToDevice, streamH));
        CHECK(hipMemcpyAsync(field->hz, deviceField->hz, bytesCount, hipMemcpyHostToDevice, streamH));

        // Spawn copy thread
        hCopyParams = (CopyParams *)malloc(sizeof(CopyParams));
        hCopyParams->xSource = field->hx;
        hCopyParams->ySource = field->hy;
        hCopyParams->zSource = field->hz;
        hCopyParams->params = params;
        hCopyParams->stream = streamH;

        hThread = (pthread_t *)malloc(sizeof(pthread_t));
        pthread_create(hThread, NULL, copyResultsWithParams, hCopyParams);

        // D field
        CHECK(hipStreamWaitEvent(streamD, eventH, 0));

        if(dThread != NULL)
            pthread_join(*dThread, NULL);

        updateDField<<<gridSize, blockSize, 0, streamD>>>(deviceField->dx2, deviceField->dy2, deviceField->dz2, 
                                                          deviceField->dx1, deviceField->dy1, deviceField->dz1, 
                                                          deviceField->hx,  deviceField->hy,  deviceField->hz);
 
        updateSources<<<gridSize, blockSize, 0, streamD>>>(deviceField->dz2, deviceField->dz1,
                                                           deviceField->hx,  deviceField->hy,
                                                           i);

        CHECK(hipEventRecord(eventD, streamD));

        CHECK(hipMemcpyAsync(field->dx0, deviceField->dx2, bytesCount, hipMemcpyDeviceToHost, streamD))
        CHECK(hipMemcpyAsync(field->dy0, deviceField->dy2, bytesCount, hipMemcpyDeviceToHost, streamD))
        CHECK(hipMemcpyAsync(field->dz0, deviceField->dz2, bytesCount, hipMemcpyDeviceToHost, streamD))

        // Spawn copy thread
        dCopyParams = (CopyParams *)malloc(sizeof(CopyParams));
        dCopyParams->xSource = field->dx0;
        dCopyParams->ySource = field->dy0;
        dCopyParams->zSource = field->dz0;
        dCopyParams->params = params;
        dCopyParams->stream = streamD;

        dThread = (pthread_t *)malloc(sizeof(pthread_t));
        pthread_create(dThread, NULL, copyResultsWithParams, dCopyParams);
            
        // E field
        CHECK(hipStreamWaitEvent(streamE, eventD, 0));

        if(eThread != NULL)
            pthread_join(*eThread, NULL);

        updateEField<<<gridSize, blockSize, 0, streamE>>>(deviceField->ex2, deviceField->ey2, deviceField->ez2, 
                                                          deviceField->ex1, deviceField->ey1, deviceField->ez1, 
                                                          deviceField->ex0, deviceField->ey0, deviceField->ez0, 
                                                          deviceField->dx2, deviceField->dy2, deviceField->dz2, 
                                                          deviceField->dx1, deviceField->dy1, deviceField->dz1, 
                                                          deviceField->dx0, deviceField->dy0, deviceField->dz0, 
                                                          deviceField->sigma, deviceField->epsI, deviceField->epsS, deviceField->tauD);
            
        updateMurBoundary<<<gridSize, blockSize, 0, streamE>>>(deviceField->ex2,  deviceField->ey2,  deviceField->ez2,                 
                                                               deviceField->ex1,  deviceField->ey1,  deviceField->ez1,                 
                                                               deviceField->rpx0, deviceField->rpy0, deviceField->rpz0,                         
                                                               deviceField->rpxEnd, deviceField->rpyEnd, deviceField->rpzEnd);

        CHECK(hipEventRecord(eventE, streamE));

        CHECK(hipMemcpyAsync(field->ex0, deviceField->ex2, bytesCount, hipMemcpyDeviceToHost, streamE))
        CHECK(hipMemcpyAsync(field->ey0, deviceField->ey2, bytesCount, hipMemcpyDeviceToHost, streamE))
        CHECK(hipMemcpyAsync(field->ez0, deviceField->ez2, bytesCount, hipMemcpyDeviceToHost, streamE))

        // Spawn copy thread
        eCopyParams = (CopyParams *)malloc(sizeof(CopyParams));
        eCopyParams->xSource = field->ex0;
        eCopyParams->ySource = field->ey0;
        eCopyParams->zSource = field->ez0;
        eCopyParams->params = params;
        eCopyParams->stream = streamD;

        eThread = (pthread_t *)malloc(sizeof(pthread_t));
        pthread_create(eThread, NULL, copyResultsWithParams, eCopyParams);

        //Spawn write results thread
        resultsParams = (ResultsParams *)malloc(sizeof(ResultsParams));
        resultsParams->params = params;
        resultsParams->field = field;
        resultsParams->hParams = hCopyParams;
        resultsParams->dParams = dCopyParams;
        resultsParams->eParams = eCopyParams;
        resultsParams->hThread = hThread;
        resultsParams->dThread = dThread;
        resultsParams->eThread = eThread;
        resultsParams->currentIteration = i+2;

        pthread_create(&threads[i+2], NULL, writeResultsWithParams, resultsParams);
    }


    // Wait for all threads to finish
    for(int i=0; i<params->iterationsCount; i++) {
        pthread_join(threads[i], NULL);
    }

    // Clean up
    //free(threads);

    deallocDeviceField(deviceField);
    deallocField(field);
    deallocParams(params);
}


FdtdParams *initParamsWithPath(const char *filePath)
{
    FdtdParams *params = (FdtdParams *)malloc(sizeof(FdtdParams));
    params->inputPath = (char *)malloc(sizeof(char) * 1024);
    params->outputPath = (char *)malloc(sizeof(char) * 1024);

    FILE *paramsFile = fopen(filePath, "r");
    //check(paramsFile != NULL, "Cannot open file");
    
    int tempLength = 1024;
    char temp[tempLength];

    //nx ny nz (field size)
    fscanf(paramsFile, "%s %d %d %d\n", temp, &params->nx, &params->ny, &params->nz);
    //t_max (simulation runs count)
    fscanf(paramsFile, "%s %d\n", temp, &params->iterationsCount);
    params->iterationsCount = ((params->iterationsCount - 1)/3 + 1) * 3; // Has to be divisible by 3
    //unused (nf)
    fgets(temp, tempLength, paramsFile);
    //env_set_dir (input path)
    fscanf(paramsFile, "%s %s\n", temp, params->inputPath);
    //unused (env_file_prefix)
    fgets(temp, tempLength, paramsFile);
    //output_dir (output path) 
    fscanf(paramsFile, "%s %s\n", temp, params->outputPath);
    //unused (output_format)
    fgets(temp, tempLength, paramsFile);
    //unused (impulse_resp_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (pec_flag) 
    fgets(temp, tempLength, paramsFile);
    //unused (read_env_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (output_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (bzip2_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (output_start)
    fgets(temp, tempLength, paramsFile);
    //unused (output_finish)
    fgets(temp, tempLength, paramsFile);
    //unused (source_type)
    fgets(temp, tempLength, paramsFile);
    //elements_per_wavelength
    fscanf(paramsFile, "%s %d\n", temp, &params->elementsPerWave);
    //wave_freq
    fscanf(paramsFile, "%s %g\n", temp, &params->waveFrequency);
    //pulse_width
    fscanf(paramsFile, "%s %g\n", temp, &params->pulseWidth);
    //pulse_modulation_frequency
    fscanf(paramsFile, "%s %g\n", temp, &params->pulseModulationFrequency);
    //number_of_excitation_sources
    fscanf(paramsFile, "%s %d\n", temp, &params->sourcesCount);
    //source_location
    params->sources = (int *)malloc(sizeof(int) * params->sourcesCount * 3);
    for(int i=0; i<params->sourcesCount; i++) {
        fscanf(paramsFile, "%s %d %d %d\n", temp,
                                            &params->sources[i*3 + 0],
                                            &params->sources[i*3 + 1],
                                            &params->sources[i*3 + 2]);
        params->sources[i*3 + 0] -= 1;
        params->sources[i*3 + 1] -= 1;
        params->sources[i*3 + 2] -= 1;
    }
    //unused (pulse_type)
    fgets(temp, tempLength, paramsFile);
    //fsigma (sigma)
    fscanf(paramsFile, "%s %f\n", temp, &params->defaultSigma);
    //feps_s (eps_s)
    fscanf(paramsFile, "%s %f\n", temp, &params->defaultEpsS);
    //feps_inf (eps_i)
    fscanf(paramsFile, "%s %f\n", temp, &params->defaultEpsI);
    //ftau_d (tau_d)
    fscanf(paramsFile, "%s %f\n", temp, &params->defaultTauD);
    
    fclose(paramsFile);

    // Generate rest of the values
    params->pi = acos(-1.0);
    params->c = 3.0 * pow(10.0, 8.0);
    params->timeskip = 1.0;
    params->lambda = params->c / params->waveFrequency;
    params->dx = params->lambda / params->elementsPerWave;
    params->dy = params->dx;
    params->dz = params->dx;
    params->dt = 1.0 * params->timeskip / (params->c * sqrt(1.0/pow(params->dx, 2.0) + 1.0/pow(params->dy, 2.0) + 1.0/pow(params->dz, 2.0)));
    params->mu0 = 4.0 * params->pi * pow(10.0, -7.0);
    params->eps0 = 1.0 / (params->mu0 * params->c * params->c);

    return params;
}


void deallocParams(FdtdParams *params)
{
    free(params->inputPath);
    free(params->outputPath);
    free(params);
}


void printParams(FdtdParams *params)
{
    printf("Field size:                 %04dx%04dx%04d\n", params->nx, params->ny, params->nz);
    printf("Iterations count:           %d\n", params->iterationsCount);
    printf("Input path:                 %s\n", params->inputPath);
    printf("Output path:                %s\n", params->outputPath);
    printf("Elements per wavelength:    %d\n", params->elementsPerWave);
    printf("Wave frequency:             %9.3E\n", params->waveFrequency);
    printf("Pulse width:                %9.3E\n", params->pulseWidth);
    printf("Pulse modulation frequency: %9.3E\n", params->pulseModulationFrequency);
    printf("Sources count:              %d\n", params->sourcesCount);
    for(int i=0; i<params->sourcesCount; i++)
        printf("Source position:            %04dx%04dx%04d\n", params->sources[i*3 + 0] + 1,
                                                               params->sources[i*3 + 1] + 1,
                                                               params->sources[i*3 + 2] + 1);
    printf("Default sigma:              %9.3E\n", params->defaultSigma);
    printf("Default eps_s:              %9.3E\n", params->defaultEpsS);
    printf("Default eps_i:              %9.3E\n", params->defaultEpsI);
    printf("Default tau_d:              %9.3E\n", params->defaultTauD);
    printf("\n");

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    printf("Running on %s\n", deviceProp.name);
    printf("Compute capability: %d.%d", deviceProp.major, deviceProp.minor);
    printf("Memory available: %.2f", deviceProp.totalGlobalMem / (1024.0 * 1024.0));

    int usedBytes = params->nx * params->ny * params->nz;
    usedBytes *= 7 * 3 + 10; // e1, e2, e3, h, d0, d1, d2, d3, eps, tau, sigma, rp, etc...
    usedBytes *= sizeof(float);
    printf("Memory requirements: %.2f MB", (float)usedBytes / (1024.0 * 1024.0));
}


FdtdField *initFieldWithParams(FdtdParams *params)
{
    int n = params->nx * params->ny * params->nz; 

    FdtdField *field = (FdtdField *)malloc(sizeof(FdtdField));
    if(field == NULL) {
        printf("Couldn't allocate field\n");
        exit(EXIT_FAILURE);
    }

    //H
    CHECK(hipHostAlloc(&field->hx, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->hy, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->hz, n * sizeof(float), hipHostMallocDefault))

    //D
    CHECK(hipHostAlloc(&field->dx0, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dy0, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dz0, n * sizeof(float), hipHostMallocDefault))

    //E
    CHECK(hipHostAlloc(&field->ex0, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ey0, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ez0, n * sizeof(float), hipHostMallocDefault))

    // sigma, eps, tau
    //CHECK(hipHostAlloc(&field->sigma, n * sizeof(float), hipHostMallocDefault))
    //CHECK(hipHostAlloc(&field->epsS,  n * sizeof(float), hipHostMallocDefault))
    //CHECK(hipHostAlloc(&field->epsI,  n * sizeof(float), hipHostMallocDefault))
    //CHECK(hipHostAlloc(&field->tauD,  n * sizeof(float), hipHostMallocDefault))
    
    field->sigma = (float *)malloc( n * sizeof(float));
    field->epsS  = (float *)malloc( n * sizeof(float));
    field->epsI  = (float *)malloc( n * sizeof(float));
    field->tauD  = (float *)malloc( n * sizeof(float));

    for(int i = 0; i < n; i++) {
        field->sigma[i] = params->defaultSigma;
        field->epsS[i]  = params->defaultEpsS;
        field->epsI[i]  = params->defaultEpsI;
        field->tauD[i]  = params->defaultTauD;
    }

    // rp
    //CHECK(hipHostAlloc(&field->rpx0, n * sizeof(float), hipHostMallocDefault))
    //CHECK(hipHostAlloc(&field->rpy0, n * sizeof(float), hipHostMallocDefault))
    //CHECK(hipHostAlloc(&field->rpz0, n * sizeof(float), hipHostMallocDefault))

    //CHECK(hipHostAlloc(&field->rpxEnd, n * sizeof(float), hipHostMallocDefault))
    //CHECK(hipHostAlloc(&field->rpyEnd, n * sizeof(float), hipHostMallocDefault))
    //CHECK(hipHostAlloc(&field->rpzEnd, n * sizeof(float), hipHostMallocDefault))

    field->rpx0 = (float *)malloc(n * sizeof(float));
    field->rpy0 = (float *)malloc(n * sizeof(float)); 
    field->rpz0 = (float *)malloc(n * sizeof(float)); 

    field->rpxEnd = (float *)malloc(n * sizeof(float));
    field->rpyEnd = (float *)malloc(n * sizeof(float));
    field->rpzEnd = (float *)malloc(n * sizeof(float));  

    return field;
}


void deallocField(FdtdField *field)
{
    //H
    CHECK(hipHostFree(field->hx));
    CHECK(hipHostFree(field->hy));
    CHECK(hipHostFree(field->hz));

    //D
    CHECK(hipHostFree(field->dx0));
    CHECK(hipHostFree(field->dy0));
    CHECK(hipHostFree(field->dz0));

    //E
    CHECK(hipHostFree(field->ex0));
    CHECK(hipHostFree(field->ey0));
    CHECK(hipHostFree(field->ez0));

    //sigma, eps, tau
    free(field->sigma);
    free(field->epsS);
    free(field->epsI);
    free(field->tauD);

    //rp
    free(field->rpx0);
    free(field->rpy0);
    free(field->rpz0);

    free(field->rpxEnd);
    free(field->rpyEnd);
    free(field->rpzEnd);

    free(field);
}


FdtdField *initDeviceFieldWithParams(FdtdParams *params)
{
    int n = params->nx * params->ny * params->nz; 

    FdtdField *field = (FdtdField *)malloc(sizeof(FdtdField));

    // e
    CHECK(hipMalloc(&field->ex0, n * sizeof(float)))
    CHECK(hipMalloc(&field->ey0, n * sizeof(float)))
    CHECK(hipMalloc(&field->ez0, n * sizeof(float)))

    CHECK(hipMalloc(&field->ex1, n * sizeof(float)))
    CHECK(hipMalloc(&field->ey1, n * sizeof(float)))
    CHECK(hipMalloc(&field->ez1, n * sizeof(float)))

    CHECK(hipMalloc(&field->ex2, n * sizeof(float)))
    CHECK(hipMalloc(&field->ey2, n * sizeof(float)))
    CHECK(hipMalloc(&field->ez2, n * sizeof(float)))

    // h
    CHECK(hipMalloc(&field->hx, n * sizeof(float)))
    CHECK(hipMalloc(&field->hy, n * sizeof(float)))
    CHECK(hipMalloc(&field->hz, n * sizeof(float)))

    // d
    CHECK(hipMalloc(&field->dx0, n * sizeof(float)))
    CHECK(hipMalloc(&field->dy0, n * sizeof(float)))
    CHECK(hipMalloc(&field->dz0, n * sizeof(float)))

    CHECK(hipMalloc(&field->dx1, n * sizeof(float)))
    CHECK(hipMalloc(&field->dy1, n * sizeof(float)))
    CHECK(hipMalloc(&field->dz1, n * sizeof(float)))

    CHECK(hipMalloc(&field->dx2, n * sizeof(float)))
    CHECK(hipMalloc(&field->dy2, n * sizeof(float)))
    CHECK(hipMalloc(&field->dz2, n * sizeof(float)))

    // sigma, eps, tau
    CHECK(hipMalloc(&field->epsI,  n * sizeof(float)))
    CHECK(hipMalloc(&field->epsS,  n * sizeof(float)))
    CHECK(hipMalloc(&field->tauD,  n * sizeof(float)))
    CHECK(hipMalloc(&field->sigma, n * sizeof(float)))

    // rp
    CHECK(hipMalloc(&field->rpx0, n * sizeof(float)))
    CHECK(hipMalloc(&field->rpy0, n * sizeof(float)))
    CHECK(hipMalloc(&field->rpz0, n * sizeof(float)))

    CHECK(hipMalloc(&field->rpxEnd, n * sizeof(float)))
    CHECK(hipMalloc(&field->rpyEnd, n * sizeof(float)))
    CHECK(hipMalloc(&field->rpzEnd, n * sizeof(float)))

    return field;
}


void deallocDeviceField(FdtdField *field)
{
    // e
    CHECK(hipFree(field->ex0))
    CHECK(hipFree(field->ey0))
    CHECK(hipFree(field->ez0))

    CHECK(hipFree(field->ex1))
    CHECK(hipFree(field->ey1))
    CHECK(hipFree(field->ez1))

    CHECK(hipFree(field->ex2))
    CHECK(hipFree(field->ey2))
    CHECK(hipFree(field->ez2))

    // h
    CHECK(hipFree(field->hx))
    CHECK(hipFree(field->hy))
    CHECK(hipFree(field->hz))

    // d
    CHECK(hipFree(field->dx0))
    CHECK(hipFree(field->dy0))
    CHECK(hipFree(field->dz0))

    CHECK(hipFree(field->dx1))
    CHECK(hipFree(field->dy1))
    CHECK(hipFree(field->dz1))

    CHECK(hipFree(field->dx2))
    CHECK(hipFree(field->dy2))
    CHECK(hipFree(field->dz2))

    // sigma, eps, tau
    CHECK(hipFree(field->epsI))
    CHECK(hipFree(field->epsS))
    CHECK(hipFree(field->tauD))
    CHECK(hipFree(field->sigma))

    // rp
    CHECK(hipFree(field->rpx0))
    CHECK(hipFree(field->rpy0))
    CHECK(hipFree(field->rpz0))

    CHECK(hipFree(field->rpxEnd))
    CHECK(hipFree(field->rpyEnd))
    CHECK(hipFree(field->rpzEnd))
}


void loadMaterials(FdtdParams *params, FdtdField *field, const char *specsFilePath, const char *materialsPath)
{
    // Load material specs
    int specsCount = 94;
    float *specs = (float *)calloc(specsCount * 4, sizeof(float));
    if(specs == NULL) {
        printf("Couldn't alocate %ld bytes for specs\n", (long)specsCount*4*sizeof(float));
        exit(EXIT_FAILURE);
    }
    char temp[1024];
    int index;
    float sigmaValue, epsSValue, epsIValue, tauDValue;

    FILE *specsFile = fopen(specsFilePath, "r");
    if(specsFile == NULL) {
        printf("Couldn\'t open file %s\n", specsFilePath);
        exit(EXIT_FAILURE);
    }

    for(int i=0; i<specsCount; i++) {
        fscanf(specsFile, "%d %s %g %g %g %g\n", &index, temp, &sigmaValue, &epsSValue, &epsIValue, &tauDValue);

        specs[index*4 + 0] = sigmaValue;
        specs[index*4 + 1] = epsSValue;
        specs[index*4 + 2] = epsIValue;
        specs[index*4 + 3] = tauDValue;

        if(index >= specsCount)
            break;
    }

    //fclose(specsFile);

    // Load materials
    for(int iz=0; iz<params->nz; iz++) {
        char materialFileName[1024];
        sprintf(materialFileName, "%s/v1_%05d.pgm", materialsPath, iz+1);
        FILE *materialFile = fopen(materialFileName, "r");
        
        if(materialFile == NULL) {
            printf("Couldn\'t open file %s\n", materialFileName);
            exit(EXIT_FAILURE);
        }

        int width, height;
        fscanf(materialFile, "%s %s %s %d %d %s", temp, temp, temp, &width, &height, temp);

        for(int iy=0; iy<params->ny; iy++) {
            for(int ix=0; ix<params->nx; ix++) {
                int code;
                fscanf(materialFile, "%d", &code);

                int offset = iz*params->nx*params->ny + iy*params->nx + ix;
                field->sigma[offset] = specs[code*4 + 0];
                field->epsS[offset] =  specs[code*4 + 1];
                field->epsI[offset] =  specs[code*4 + 2];
                field->tauD[offset] =  specs[code*4 + 3];
            }
        }

        fclose(materialFile);
    }

    //free(specs);
}


void setupMurBoundary(FdtdParams *params, FdtdField *field)
{
#ifndef __APPLE__
    int nx = params->nx;
    int ny = params->ny;
    int nz = params->nz;

    // Setup rpx
    for(int iz = 0; iz < nz; iz++) {
        for(int iy = 0; iy < ny; iy++) {
            for(int ix = 0; ix < 2; ix++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy,iz);
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) / (2.0 * params->pi * params->waveFrequency * params->eps0);

                OFFSET(field->rpx0, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +
                                                        (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }

            for(int ix = nx - 2; ix < nx; ix++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy, iz);
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) / (2.0 * params->pi * params->waveFrequency * params->eps0);
                
                OFFSET(field->rpxEnd, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +                                                  
                                                          (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }
        }
    }

    // Setup rpy
    for(int iz = 0; iz < nz; iz++) {
        for(int ix = 0; ix < nx; ix++) {
            for(int iy = 0; iy < 2; iy++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy, iz) * I;
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) /(2.0 * params->pi * params->waveFrequency * params->eps0) * I;
                
                OFFSET(field->rpy0, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +                                                      
                                                        (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }

            for(int iy = ny - 2; iy < ny; iy++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy, iz) * I;
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) / (2 * params->pi * params->waveFrequency * params->eps0) * I;
                
                OFFSET(field->rpyEnd, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +                                                      
                                                          (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }
        }
    }

    // Setup rpz
    for(int iy = 0; iy < ny; iy++) {
        for(int ix = 0; ix < nx; ix++) {
            for(int iz = 0; iz < 2; iz++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy, iz) * I;
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) / (2.0 * params->pi * params->waveFrequency * params->eps0) * I;
                
                OFFSET(field->rpz0, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +                                                  
                                                        (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }

            for(int iz = nz - 2; iz < nz; iz++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy, iz) * I;
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) / (2.0 * params->pi * params->waveFrequency * params->eps0) * I;
                
                OFFSET(field->rpzEnd, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +                                                  
                                                          (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }
        }
    }
#endif
}


void setupSources(FdtdParams *params)
{
    int fine, temp, i2, istart;
    float *tmpdata, *tmpdata2;
    int tmpOff = 1<<16;

    params->jzCount = tmpOff;
    params->jz = (float *)calloc(tmpOff,     sizeof(float));
    tmpdata    = (float *)calloc(tmpOff * 2, sizeof(float));
    tmpdata2   = (float *)calloc(tmpOff * 2, sizeof(float));
    
    //fine & temp
    fine = (1<<13) * params->pulseWidth * params->waveFrequency * params->dt;
    temp = 1.0/(params->pulseWidth * params->waveFrequency)/(params->dt / fine)/2.0;
    
    //tmpdata
    for(int i = -temp - 1; i <= temp + 1; i++) {
        float v1 = ((float)i/(((float)temp + 1.0)/4.0));
        float v2 = exp(-pow(v1, 2.0));
        float v3 = cos(2.0 * acos(-1.0) * params->pulseModulationFrequency * params->waveFrequency * i * (params->dt / fine));
        tmpdata[i + tmpOff] = v2 * v3;
    }

    //istart
    for(int i = -(1<<12); i < (1<<12); i++) {
         if((fabs(tmpdata[i + tmpOff]) >= 1e-9) && (i % fine == 0)) {
            istart = i;
            break;
         }
    }
    
    //setup jz 1/2
    i2 = 0;
    for(int i = istart; i <= temp+1; i += fine) {
        float val = tmpdata[i + tmpOff] * 1e-15 / params->dt / 3.0;
        params->jz[i2] = val;
        i2++;
    }
    
    //setup tmpdata2
    for(int i = 2; i <= (1<<14); i++) {
        float val = (((params->jz[i + 1 - 1] - params->jz[i - 1]) / params->dt) +
                     ((params->jz[i - 1] - params->jz[i - 1 - 1]) / params->dt)) / 
                    2.0 * (params->dt * params->dz) / (params->dx * params->dy * params->dz);
                                    
        tmpdata2[i - 1 + tmpOff] = val;
    }
    
    //setup jz 2/2
    for(int i=0; i < 1<<14; i++) {
        params->jz[i] = tmpdata2[i + tmpOff + 1];
    }

    free(tmpdata2);
    free(tmpdata);
}


void copyDataToDevice(FdtdParams *params, FdtdField *field, FdtdField *deviceField)
{
    int bytesCount = params->nx * params->ny * params->nz * sizeof(float); 

    //H
    CHECK(hipMemset(deviceField->hx, 0, bytesCount))
    CHECK(hipMemset(deviceField->hy, 0, bytesCount))
    CHECK(hipMemset(deviceField->hz, 0, bytesCount))

    //D
    CHECK(hipMemset(deviceField->dx0, 0, bytesCount))
    CHECK(hipMemset(deviceField->dy0, 0, bytesCount))
    CHECK(hipMemset(deviceField->dz0, 0, bytesCount))

    CHECK(hipMemset(deviceField->dx1, 0, bytesCount))
    CHECK(hipMemset(deviceField->dy1, 0, bytesCount))
    CHECK(hipMemset(deviceField->dz1, 0, bytesCount))

    CHECK(hipMemset(deviceField->dx2, 0, bytesCount))
    CHECK(hipMemset(deviceField->dy2, 0, bytesCount))
    CHECK(hipMemset(deviceField->dz2, 0, bytesCount))

    //E
    CHECK(hipMemset(deviceField->ex0, 0, bytesCount))
    CHECK(hipMemset(deviceField->ey0, 0, bytesCount))
    CHECK(hipMemset(deviceField->ez0, 0, bytesCount))

    CHECK(hipMemset(deviceField->ex1, 0, bytesCount))
    CHECK(hipMemset(deviceField->ey1, 0, bytesCount))
    CHECK(hipMemset(deviceField->ez1, 0, bytesCount))

    CHECK(hipMemset(deviceField->ex2, 0, bytesCount))
    CHECK(hipMemset(deviceField->ey2, 0, bytesCount))
    CHECK(hipMemset(deviceField->ez2, 0, bytesCount))

    //eps, tau, sigma
    CHECK(hipMemcpy(deviceField->epsI,  field->epsI,  bytesCount, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->epsS,  field->epsS,  bytesCount, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->tauD,  field->tauD,  bytesCount, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->sigma, field->sigma, bytesCount, hipMemcpyHostToDevice))

    CHECK(hipMemcpy(deviceField->rpx0, field->rpx0, bytesCount, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->rpy0, field->rpy0, bytesCount, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->rpz0, field->rpz0, bytesCount, hipMemcpyHostToDevice))

    CHECK(hipMemcpy(deviceField->rpxEnd, field->rpxEnd, bytesCount, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->rpyEnd, field->rpyEnd, bytesCount, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->rpzEnd, field->rpzEnd, bytesCount, hipMemcpyHostToDevice))
}


void *copyResultsWithParams(void *params)
{
    CopyParams *copyParams = (CopyParams *)params;

    int bytesCount = copyParams->params->nx * copyParams->params->ny * copyParams->params->nz * sizeof(float);

    copyParams->xBuffer = (float *)malloc(bytesCount);
    copyParams->yBuffer = (float *)malloc(bytesCount);
    copyParams->zBuffer = (float *)malloc(bytesCount);

    CHECK(hipStreamSynchronize(copyParams->stream))

    memcpy(copyParams->xBuffer, copyParams->xSource, bytesCount);
    memcpy(copyParams->yBuffer, copyParams->ySource, bytesCount);
    memcpy(copyParams->zBuffer, copyParams->zSource, bytesCount);

    pthread_exit(NULL);
}


void *writeResultsWithParams(void *params)
{
    ResultsParams *resultsParams = (ResultsParams *)params;

    pthread_join(*resultsParams->hThread, NULL);
    pthread_join(*resultsParams->dThread, NULL);
    pthread_join(*resultsParams->eThread, NULL);

    writeResults(resultsParams->params, resultsParams->field,
                 resultsParams->hParams->xBuffer, resultsParams->hParams->yBuffer, resultsParams->hParams->zBuffer,
                 resultsParams->dParams->xBuffer, resultsParams->dParams->yBuffer, resultsParams->dParams->zBuffer,
                 resultsParams->eParams->xBuffer, resultsParams->eParams->yBuffer, resultsParams->eParams->zBuffer,
                 resultsParams->currentIteration);

    free(resultsParams->hParams);
    free(resultsParams->dParams);
    free(resultsParams->eParams);

    free(params);

    pthread_exit(NULL);
}


void writeResults(FdtdParams *params, FdtdField *field,
                  float *hxSource, float *hySource, float *hzSource,
                  float *dxSource, float *dySource, float *dzSource,
                  float *exSource, float *eySource, float *ezSource,
                  int currentIteration)
{
    char outputFilePath[1024];
    FILE *outputFile;

    // Used by OFFSET macro
    int nx = params->nx;
    int ny = params->ny;
    int nz = params->nz;

    // Output hx
    for(int iz = 0; iz < params->nz; iz++) {
        sprintf(outputFilePath, "%s/H_field_x_%05d_z%05d.out", params->outputPath, currentIteration, iz);

        outputFile = fopen(outputFilePath, "w");
        if(outputFile == NULL) {
            printf("Couldn\'t open file %s\n", outputFilePath);
            exit(EXIT_FAILURE);
        }

        for(int iy = 0; iy < params->ny; iy++) {
            for(int ix=0; ix < params->nx; ix++)
                fprintf(outputFile, "% 9.3E", OFFSET(hxSource, ix, iy, iz));
            
            fprintf(outputFile, "\n");
        }

        fclose(outputFile);
    }

    // Output hy
    for(int iz = 0; iz < params->nz; iz++) {
        sprintf(outputFilePath, "%s/H_field_y_%05d_z%05d.out", params->outputPath, currentIteration, iz);

        outputFile = fopen(outputFilePath, "w");
        if(outputFile == NULL) {
            printf("Couldn\'t open file %s\n", outputFilePath);
            exit(EXIT_FAILURE);
        }

        for(int iy = 0; iy < params->ny; iy++) {
            for(int ix=0; ix < params->nx; ix++)
                fprintf(outputFile, "% 9.3E", OFFSET(hySource, ix, iy, iz));
            
            fprintf(outputFile, "\n");
        }

        fclose(outputFile);
    }

    // Output hz
    for(int iz = 0; iz < params->nz; iz++) {
        sprintf(outputFilePath, "%s/H_field_z_%05d_z%05d.out", params->outputPath, currentIteration, iz);

        outputFile = fopen(outputFilePath, "w");
        if(outputFile == NULL) {
            printf("Couldn\'t open file %s\n", outputFilePath);
            exit(EXIT_FAILURE);
        }

        for(int iy = 0; iy < params->ny; iy++) {
            for(int ix=0; ix < params->nx; ix++)
                fprintf(outputFile, "% 9.3E", OFFSET(hzSource, ix, iy, iz));
            
            fprintf(outputFile, "\n");
        }

        fclose(outputFile);
    }

    // Output dx
    for(int iz = 0; iz < params->nz; iz++) {
        sprintf(outputFilePath, "%s/D_field_x_%05d_z%05d.out", params->outputPath, currentIteration, iz);

        outputFile = fopen(outputFilePath, "w");
        if(outputFile == NULL) {
            printf("Couldn\'t open file %s\n", outputFilePath);
            exit(EXIT_FAILURE);
        }

        for(int iy = 0; iy < params->ny; iy++) {
            for(int ix=0; ix < params->nx; ix++)
                fprintf(outputFile, "% 9.3E", OFFSET(dxSource, ix, iy, iz));
            
            fprintf(outputFile, "\n");
        }

        fclose(outputFile);
    }

    // Output dy
    for(int iz = 0; iz < params->nz; iz++) {
        sprintf(outputFilePath, "%s/D_field_y_%05d_z%05d.out", params->outputPath, currentIteration, iz);

        outputFile = fopen(outputFilePath, "w");
        if(outputFile == NULL) {
            printf("Couldn\'t open file %s\n", outputFilePath);
            exit(EXIT_FAILURE);
        }

        for(int iy = 0; iy < params->ny; iy++) {
            for(int ix=0; ix < params->nx; ix++)
                fprintf(outputFile, "% 9.3E", OFFSET(dySource, ix, iy, iz));
            
            fprintf(outputFile, "\n");
        }

        fclose(outputFile);
    }

    // Output dz
    for(int iz = 0; iz < params->nz; iz++) {
        sprintf(outputFilePath, "%s/D_field_z_%05d_z%05d.out", params->outputPath, currentIteration, iz);

        outputFile = fopen(outputFilePath, "w");
        if(outputFile == NULL) {
            printf("Couldn\'t open file %s\n", outputFilePath);
            exit(EXIT_FAILURE);
        }

        for(int iy = 0; iy < params->ny; iy++) {
            for(int ix=0; ix < params->nx; ix++)
                fprintf(outputFile, "% 9.3E", OFFSET(dzSource, ix, iy, iz));
            
            fprintf(outputFile, "\n");
        }

        fclose(outputFile);
    }

    // Output ex
    for(int iz = 0; iz < params->nz; iz++) {
        sprintf(outputFilePath, "%s/E_field_x_%05d_z%05d.out", params->outputPath, currentIteration, iz);

        outputFile = fopen(outputFilePath, "w");
        if(outputFile == NULL) {
            printf("Couldn\'t open file %s\n", outputFilePath);
            exit(EXIT_FAILURE);
        }

        for(int iy = 0; iy < params->ny; iy++) {
            for(int ix=0; ix < params->nx; ix++)
                fprintf(outputFile, "% 9.3E", OFFSET(exSource, ix, iy, iz));
            
            fprintf(outputFile, "\n");
        }

        fclose(outputFile);
    }

    // Output ey
    for(int iz = 0; iz < params->nz; iz++) {
        sprintf(outputFilePath, "%s/E_field_y_%05d_z%05d.out", params->outputPath, currentIteration, iz);

        outputFile = fopen(outputFilePath, "w");
        if(outputFile == NULL) {
            printf("Couldn\'t open file %s\n", outputFilePath);
            exit(EXIT_FAILURE);
        }

        for(int iy = 0; iy < params->ny; iy++) {
            for(int ix=0; ix < params->nx; ix++)
                fprintf(outputFile, "% 9.3E", OFFSET(eySource, ix, iy, iz));
            
            fprintf(outputFile, "\n");
        }

        fclose(outputFile);
    }

    // Output ez
    for(int iz = 0; iz < params->nz; iz++) {
        sprintf(outputFilePath, "%s/E_field_z_%05d_z%05d.out", params->outputPath, currentIteration, iz);

        outputFile = fopen(outputFilePath, "w");
        if(outputFile == NULL) {
            printf("Couldn\'t open file %s\n", outputFilePath);
            exit(EXIT_FAILURE);
        }

        for(int iy = 0; iy < params->ny; iy++) {
            for(int ix=0; ix < params->nx; ix++)
                fprintf(outputFile, "% 9.3E", OFFSET(ezSource, ix, iy, iz));
            
            fprintf(outputFile, "\n");
        }

        fclose(outputFile);
    }


    //Cleanup unnecessary buffers
    free(hxSource);
    free(hySource);
    free(hzSource);

    free(dxSource);
    free(dySource);
    free(dzSource);

    free(exSource);
    free(eySource);
    free(ezSource);
}
