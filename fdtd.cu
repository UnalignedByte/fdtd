#include "hip/hip_runtime.h"
#include "fdtd.h"

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>

#include "utils.h"
#include "fdtd_calculations.h"


#define BLOCK_X 4
#define BLOCK_Y 4
#define BLOCK_Z 4


int main(int argc, char **argv)
{
    // Read params
    FdtdParams *params;
    printf("Reading parameters...\n");
    params = initParamsWithPath("data/input_params");
    printParams(params);

    // Initialize field
    FdtdField  *field, *deviceField; // Used for CUDA

    printf("Initializing field...\n");
    field = initFieldWithParams(params);
    setupMurBoundary(params, field);

    printf("Initializing device field...\n");
    deviceField = initDeviceFieldWithParams(params);

    printf("Reading materials data...\n");
    loadMaterials(params, field, "data/mat_specs_riken", params->inputPath);

    printf("Initializing sources...\n");
    setupSources(params);

    printf("Copying data to GPU...\n");
    copyData(params, field, deviceField);

    // Copy array params to device
    float *deviceJz;
    int *deviceSources;
    int bytesCount;

    bytesCount = (1<<16) * sizeof(float);
    CHECK(hipMalloc(&deviceJz, bytesCount))
    CHECK(hipMemcpy(deviceJz, params->jz, bytesCount, hipMemcpyHostToDevice))

    bytesCount = params->sourcesCount * sizeof(int);
    CHECK(hipMalloc(&deviceSources, bytesCount))
    CHECK(hipMemcpy(deviceSources, params->sources, bytesCount, hipMemcpyHostToDevice))

    // Setup CUDA parameters
    dim3 gridSize = dim3((params->nx + BLOCK_X - 1)/BLOCK_X,
                         (params->ny + BLOCK_Y - 1)/BLOCK_Y,
                         (params->nz + BLOCK_Z - 1)/BLOCK_Z);
    dim3 blockSize = dim3(BLOCK_X, BLOCK_Y, BLOCK_Z);

    // Main loop
    for(int i=0; i<params->iterationsCount; i += 3) {
        // Run 0
        printf("Running iteration %d\n", i);

        updateHField<<<gridSize, blockSize>>>(deviceField->hx,  deviceField->hy,  deviceField->hz,                    
                                              deviceField->ex2, deviceField->ey2, deviceField->ez2,                 
                                              params->nx, params->ny, params->nz,                 
                                              params->dt, params->dx, params->dy, params->dz, 
                                              params->mu0);
        CHECK(hipDeviceSynchronize())

        updateDField<<<gridSize, blockSize>>>(deviceField->dx0, deviceField->dy0, deviceField->dz0, 
                                              deviceField->dx2, deviceField->dy2, deviceField->dz2, 
                                              deviceField->hx,  deviceField->hy,  deviceField->hz,    
                                              params->nx, params->ny, params->nz, 
                                              params->dt, params->dx, params->dy, params->dz);
        CHECK(hipDeviceSynchronize())
 
        updateSources<<<gridSize, blockSize>>>(deviceField->dz0, deviceField->dz2,                                 
                                               deviceField->hx,  deviceField->hy,                                   
                                               params->nx, params->ny, params->nz,
                                               params->dt, params->dx, params->dy, params->dz, 
                                               deviceSources, deviceJz,                                
                                               params->sourcesCount, i);
        CHECK(hipDeviceSynchronize())
            
        updateEField<<<gridSize, blockSize>>>(deviceField->ex0, deviceField->ey0, deviceField->ez0, 
                                              deviceField->ex2, deviceField->ey2, deviceField->ez2, 
                                              deviceField->ex1, deviceField->ey1, deviceField->ez1, 
                                              deviceField->dx0, deviceField->dy0, deviceField->dz0, 
                                              deviceField->dx2, deviceField->dy2, deviceField->dz2, 
                                              deviceField->dx1, deviceField->dy1, deviceField->dz1, 
                                              deviceField->sigma, deviceField->epsI, deviceField->epsS, deviceField->tauD,             
                                              params->nx, params->ny, params->nz, 
                                              params->dt, params->eps0);
        CHECK(hipDeviceSynchronize())
            
        updateMurBoundary<<<gridSize, blockSize>>>(deviceField->ex0,  deviceField->ey0,  deviceField->ez0,                 
                                                   deviceField->ex2,  deviceField->ey2,  deviceField->ez2,                 
                                                   deviceField->rpx0, deviceField->rpy0, deviceField->rpz0,                         
                                                   deviceField->rpxEnd, deviceField->rpyEnd, deviceField->rpzEnd,                         
                                                   params->nx, params->ny, params->nz,                 
                                                   params->dt, params->dx, params->dy, params->dz, 
                                                   params->mu0, params->eps0);
        CHECK(hipDeviceSynchronize());

        // Write results
        writeResults(params, field,
                     field->ex0, field->ey0, field->ez0,
                     field->dx0, field->dy0, field->dz0,
                     i, params->outputPath);

        // Run 1
        printf("Running iteration %d\n", i+1);

        updateHField<<<gridSize, blockSize>>>(deviceField->hx,  deviceField->hy,  deviceField->hz,                    
                                              deviceField->ex0, deviceField->ey0, deviceField->ez0,                 
                                              params->nx, params->ny, params->nz,                 
                                              params->dt, params->dx, params->dy, params->dz, 
                                              params->mu0);
        CHECK(hipDeviceSynchronize())

        updateDField<<<gridSize, blockSize>>>(deviceField->dx1, deviceField->dy1, deviceField->dz1, 
                                              deviceField->dx0, deviceField->dy0, deviceField->dz0, 
                                              deviceField->hx,  deviceField->hy,  deviceField->hz,    
                                              params->nx, params->ny, params->nz, 
                                              params->dt, params->dx, params->dy, params->dz);
        CHECK(hipDeviceSynchronize())
 
        updateSources<<<gridSize, blockSize>>>(deviceField->dz1, deviceField->dz0,                                 
                                               deviceField->hx,  deviceField->hy,                                   
                                               params->nx, params->ny, params->nz,
                                               params->dt, params->dx, params->dy, params->dz, 
                                               deviceSources, deviceJz,                                
                                               params->sourcesCount, i);
        CHECK(hipDeviceSynchronize())
            
        updateEField<<<gridSize, blockSize>>>(deviceField->ex1, deviceField->ey1, deviceField->ez1, 
                                              deviceField->ex0, deviceField->ey0, deviceField->ez0, 
                                              deviceField->ex2, deviceField->ey2, deviceField->ez2, 
                                              deviceField->dx1, deviceField->dy1, deviceField->dz1, 
                                              deviceField->dx0, deviceField->dy0, deviceField->dz0, 
                                              deviceField->dx2, deviceField->dy2, deviceField->dz2, 
                                              deviceField->sigma, deviceField->epsI, deviceField->epsS, deviceField->tauD,             
                                              params->nx, params->ny, params->nz, 
                                              params->dt, params->eps0);
        CHECK(hipDeviceSynchronize())
            
        updateMurBoundary<<<gridSize, blockSize>>>(deviceField->ex1,  deviceField->ey1,  deviceField->ez1,                 
                                                   deviceField->ex0,  deviceField->ey0,  deviceField->ez0,                 
                                                   deviceField->rpx0, deviceField->rpy0, deviceField->rpz0,                         
                                                   deviceField->rpxEnd, deviceField->rpyEnd, deviceField->rpzEnd,                         
                                                   params->nx, params->ny, params->nz,                 
                                                   params->dt, params->dx, params->dy, params->dz, 
                                                   params->mu0, params->eps0);
        CHECK(hipDeviceSynchronize())

        // Write results
        writeResults(params, field,
                     field->ex1, field->ey1, field->ez1,
                     field->dx1, field->dy1, field->dz1,
                     i+1, params->outputPath);

        // Run 2
        printf("Running iteration %d\n", i+2);

        updateHField<<<gridSize, blockSize>>>(deviceField->hx,  deviceField->hy,  deviceField->hz,                    
                                              deviceField->ex1, deviceField->ey1, deviceField->ez1,                 
                                              params->nx, params->ny, params->nz,                 
                                              params->dt, params->dx, params->dy, params->dz, 
                                              params->mu0);
        CHECK(hipDeviceSynchronize())

        updateDField<<<gridSize, blockSize>>>(deviceField->dx2, deviceField->dy2, deviceField->dz2, 
                                              deviceField->dx1, deviceField->dy1, deviceField->dz1, 
                                              deviceField->hx,  deviceField->hy,  deviceField->hz,    
                                              params->nx, params->ny, params->nz, 
                                              params->dt, params->dx, params->dy, params->dz);
        CHECK(hipDeviceSynchronize())
 
        updateSources<<<gridSize, blockSize>>>(deviceField->dz2, deviceField->dz1,                                 
                                               deviceField->hx,  deviceField->hy,                                   
                                               params->nx, params->ny, params->nz,
                                               params->dt, params->dx, params->dy, params->dz, 
                                               deviceSources, deviceJz,                                
                                               params->sourcesCount, i);
        CHECK(hipDeviceSynchronize())
            
        updateEField<<<gridSize, blockSize>>>(deviceField->ex2, deviceField->ey2, deviceField->ez2, 
                                              deviceField->ex1, deviceField->ey1, deviceField->ez1, 
                                              deviceField->ex0, deviceField->ey0, deviceField->ez0, 
                                              deviceField->dx2, deviceField->dy2, deviceField->dz2, 
                                              deviceField->dx1, deviceField->dy1, deviceField->dz1, 
                                              deviceField->dx0, deviceField->dy0, deviceField->dz0, 
                                              deviceField->sigma, deviceField->epsI, deviceField->epsS, deviceField->tauD,             
                                              params->nx, params->ny, params->nz, 
                                              params->dt, params->eps0);
        CHECK(hipDeviceSynchronize())
            
        updateMurBoundary<<<gridSize, blockSize>>>(deviceField->ex2,  deviceField->ey2,  deviceField->ez2,                 
                                                   deviceField->ex1,  deviceField->ey1,  deviceField->ez1,                 
                                                   deviceField->rpx0, deviceField->rpy0, deviceField->rpz0,                         
                                                   deviceField->rpxEnd, deviceField->rpyEnd, deviceField->rpzEnd,                         
                                                   params->nx, params->ny, params->nz,                 
                                                   params->dt, params->dx, params->dy, params->dz, 
                                                   params->mu0, params->eps0);
        CHECK(hipDeviceSynchronize())

        // Write results
        writeResults(params, field,
                     field->ex2, field->ey2, field->ez2,
                     field->dx2, field->dy2, field->dz2,
                     i+2, params->outputPath);
    }

    // Clean up
    /*deallocDeviceField(deviceField);
    deallocField(field);
    deallocParams(params);*/
}


FdtdParams *initParamsWithPath(const char *filePath)
{
    FdtdParams *params = (FdtdParams *)malloc(sizeof(FdtdParams));
    params->inputPath = (char *)malloc(sizeof(char) * 1024);
    params->outputPath = (char *)malloc(sizeof(char) * 1024);

    FILE *paramsFile = fopen(filePath, "r");
    //check(paramsFile != NULL, "Cannot open file");
    
    int tempLength = 1024;
    char temp[tempLength];

    //nx ny nz (field size)
    fscanf(paramsFile, "%s %d %d %d\n", temp, &params->nx, &params->ny, &params->nz);
    //t_max (simulation runs count)
    fscanf(paramsFile, "%s %d\n", temp, &params->iterationsCount);
    params->iterationsCount = ((params->iterationsCount - 1)/3 + 1) * 3; // Has to be divisible by 3
    //unused (nf)
    fgets(temp, tempLength, paramsFile);
    //env_set_dir (input path)
    fscanf(paramsFile, "%s %s\n", temp, params->inputPath);
    //unused (env_file_prefix)
    fgets(temp, tempLength, paramsFile);
    //output_dir (output path) 
    fscanf(paramsFile, "%s %s\n", temp, params->outputPath);
    //unused (output_format)
    fgets(temp, tempLength, paramsFile);
    //unused (impulse_resp_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (pec_flag) 
    fgets(temp, tempLength, paramsFile);
    //unused (read_env_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (output_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (bzip2_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (output_start)
    fgets(temp, tempLength, paramsFile);
    //unused (output_finish)
    fgets(temp, tempLength, paramsFile);
    //unused (source_type)
    fgets(temp, tempLength, paramsFile);
    //elements_per_wavelength
    fscanf(paramsFile, "%s %d\n", temp, &params->elementsPerWave);
    //wave_freq
    fscanf(paramsFile, "%s %g\n", temp, &params->waveFrequency);
    //pulse_width
    fscanf(paramsFile, "%s %g\n", temp, &params->pulseWidth);
    //pulse_modulation_frequency
    fscanf(paramsFile, "%s %g\n", temp, &params->pulseModulationFrequency);
    //number_of_excitation_sources
    fscanf(paramsFile, "%s %d\n", temp, &params->sourcesCount);
    //source_location
    params->sources = (int *)malloc(sizeof(int) * params->sourcesCount * 3);
    for(int i=0; i<params->sourcesCount; i++) {
        fscanf(paramsFile, "%s %d %d %d\n", temp,
                                            &params->sources[i*3],
                                            &params->sources[i*3 + 1],
                                            &params->sources[i*3 + 2]);
    }
    //unused (pulse_type)
    fgets(temp, tempLength, paramsFile);
    //fsigma (sigma)
    fscanf(paramsFile, "%s %f\n", temp, &params->defaultSigma);
    //feps_s (eps_s)
    fscanf(paramsFile, "%s %f\n", temp, &params->defaultEpsS);
    //feps_inf (eps_i)
    fscanf(paramsFile, "%s %f\n", temp, &params->defaultEpsI);
    //ftau_d (tau_d)
    fscanf(paramsFile, "%s %f\n", temp, &params->defaultTauD);
    
    fclose(paramsFile);

    // Generate rest of the values
    params->pi = acos(-1.0);
    params->c = 3.0 * pow(10.0, 8.0);
    params->timeskip = 1.0;
    params->lambda = params->c / params->waveFrequency;
    params->dx = params->lambda / params->elementsPerWave;
    params->dy = params->dx;
    params->dz = params->dx;
    params->dt = 1.0 * params->timeskip / (params->c * sqrt(1.0/pow(params->dx, 2.0) + 1.0/pow(params->dy, 2.0) + 1.0/pow(params->dz, 2.0)));
    params->mu0 = 4.0 * params->pi * pow(10.0, -7.0);
    params->eps0 = 1.0 / params->mu0 * (params->c * params->c);

    return params;
}


void deallocParams(FdtdParams *params)
{
    free(params->inputPath);
    free(params->outputPath);
    free(params);
}


void printParams(FdtdParams *params)
{
    printf("Field size:                 %dx%dx%d\n", params->nx, params->ny, params->nz);
    printf("Iterations count:           %d\n", params->iterationsCount);
    printf("Input path:                 %s\n", params->inputPath);
    printf("Output path:                %s\n", params->outputPath);
    printf("Elements per wavelength:    %d\n", params->elementsPerWave);
    printf("Wave frequency:             %g\n", params->waveFrequency);
    printf("Pulse width:                %g\n", params->pulseWidth);
    printf("Pulse modulation frequency: %g\n", params->pulseModulationFrequency);
    printf("Sources count:              %d\n", params->sourcesCount);
    for(int i=0; i<params->sourcesCount; i++)
        printf("Source position:            %dx%dx%d\n", params->sources[i*3],
                                                         params->sources[i*3 + 1],
                                                         params->sources[i*3 + 2]);
    printf("Default sigma:              %g\n", params->defaultSigma);
    printf("Default eps_s:              %g\n", params->defaultEpsS);
    printf("Default eps_i:              %g\n", params->defaultEpsI);
    printf("Default tau_d:              %g\n", params->defaultTauD);
    printf("\n");
}


FdtdField *initFieldWithParams(FdtdParams *params)
{
    int n = params->nx * params->ny * params->nz; 

    FdtdField *field = (FdtdField *)malloc(sizeof(FdtdField));
    if(field == NULL) {
        printf("Couldn't allocate field\n");
        exit(EXIT_FAILURE);
    }

    // e
    CHECK(hipHostAlloc(&field->ex0, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ey0, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ez0, n * sizeof(float), hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->ex1, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ey1, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ez1, n * sizeof(float), hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->ex2, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ey2, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ez2, n * sizeof(float), hipHostMallocDefault))

    // h
    CHECK(hipHostAlloc(&field->hx, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->hy, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->hz, n * sizeof(float), hipHostMallocDefault))

    // d
    CHECK(hipHostAlloc(&field->dx0, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dy0, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dz0, n * sizeof(float), hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->dx1, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dy1, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dz1, n * sizeof(float), hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->dx2, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dy2, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dz2, n * sizeof(float), hipHostMallocDefault))

    // sigma, eps, tau
    CHECK(hipHostAlloc(&field->sigma, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->epsS,  n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->epsI,  n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->tauD,  n * sizeof(float), hipHostMallocDefault))

    for(int i = 0; i < n; i++) {
        field->sigma[i] = params->defaultSigma;
        field->epsS[i]  = params->defaultEpsS;
        field->epsI[i]  = params->defaultEpsI;
        field->tauD[i]  = params->defaultTauD;
    }

    // rp
    CHECK(hipHostAlloc(&field->rpx0, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->rpy0, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->rpz0, n * sizeof(float), hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->rpxEnd, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->rpyEnd, n * sizeof(float), hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->rpzEnd, n * sizeof(float), hipHostMallocDefault))

    return field;
}


void deallocField(FdtdField *field)
{
    // e
    CHECK(hipFree(field->ex0))
    CHECK(hipFree(field->ey0))
    CHECK(hipFree(field->ez0))

    CHECK(hipFree(field->ex1))
    CHECK(hipFree(field->ey1))
    CHECK(hipFree(field->ez1))

    CHECK(hipFree(field->ex2))
    CHECK(hipFree(field->ey2))
    CHECK(hipFree(field->ez2))

    // h
    CHECK(hipFree(field->hx))
    CHECK(hipFree(field->hy))
    CHECK(hipFree(field->hz))

    // d
    CHECK(hipFree(field->dx0))
    CHECK(hipFree(field->dy0))
    CHECK(hipFree(field->dz0))

    CHECK(hipFree(field->dx1))
    CHECK(hipFree(field->dy1))
    CHECK(hipFree(field->dz1))

    CHECK(hipFree(field->dx2))
    CHECK(hipFree(field->dy2))
    CHECK(hipFree(field->dz2))

    // sigma, eps, tau
    CHECK(hipFree(&field->sigma))
    CHECK(hipFree(&field->epsS))
    CHECK(hipFree(&field->epsI))
    CHECK(hipFree(&field->tauD))

    // rp
    CHECK(hipFree(&field->rpx0))
    CHECK(hipFree(&field->rpy0))
    CHECK(hipFree(&field->rpz0))

    CHECK(hipFree(&field->rpxEnd))
    CHECK(hipFree(&field->rpyEnd))
    CHECK(hipFree(&field->rpzEnd))

    free(field);
}


FdtdField *initDeviceFieldWithParams(FdtdParams *params)
{
    int n = params->nx * params->ny * params->nz; 

    FdtdField *field = (FdtdField *)malloc(sizeof(FdtdField));

    // e
    CHECK(hipMalloc(&field->ex0, n * sizeof(float)))
    CHECK(hipMalloc(&field->ey0, n * sizeof(float)))
    CHECK(hipMalloc(&field->ez0, n * sizeof(float)))

    CHECK(hipMalloc(&field->ex1, n * sizeof(float)))
    CHECK(hipMalloc(&field->ey1, n * sizeof(float)))
    CHECK(hipMalloc(&field->ez1, n * sizeof(float)))

    CHECK(hipMalloc(&field->ex2, n * sizeof(float)))
    CHECK(hipMalloc(&field->ey2, n * sizeof(float)))
    CHECK(hipMalloc(&field->ez2, n * sizeof(float)))

    // h
    CHECK(hipMalloc(&field->hx, n * sizeof(float)))
    CHECK(hipMalloc(&field->hy, n * sizeof(float)))
    CHECK(hipMalloc(&field->hz, n * sizeof(float)))

    // d
    CHECK(hipMalloc(&field->dx0, n * sizeof(float)))
    CHECK(hipMalloc(&field->dy0, n * sizeof(float)))
    CHECK(hipMalloc(&field->dz0, n * sizeof(float)))

    CHECK(hipMalloc(&field->dx1, n * sizeof(float)))
    CHECK(hipMalloc(&field->dy1, n * sizeof(float)))
    CHECK(hipMalloc(&field->dz1, n * sizeof(float)))

    CHECK(hipMalloc(&field->dx2, n * sizeof(float)))
    CHECK(hipMalloc(&field->dy2, n * sizeof(float)))
    CHECK(hipMalloc(&field->dz2, n * sizeof(float)))

    // sigma, eps, tau
    CHECK(hipMalloc(&field->epsI,  n * sizeof(float)))
    CHECK(hipMalloc(&field->epsS,  n * sizeof(float)))
    CHECK(hipMalloc(&field->tauD,  n * sizeof(float)))
    CHECK(hipMalloc(&field->sigma, n * sizeof(float)))

    // rp
    CHECK(hipMalloc(&field->rpx0, n * sizeof(float)))
    CHECK(hipMalloc(&field->rpy0, n * sizeof(float)))
    CHECK(hipMalloc(&field->rpz0, n * sizeof(float)))

    CHECK(hipMalloc(&field->rpxEnd, n * sizeof(float)))
    CHECK(hipMalloc(&field->rpyEnd, n * sizeof(float)))
    CHECK(hipMalloc(&field->rpzEnd, n * sizeof(float)))

    return field;
}


void deallocDeviceField(FdtdField *field)
{
    // e
    CHECK(hipFree(field->ex0))
    CHECK(hipFree(field->ey0))
    CHECK(hipFree(field->ez0))

    CHECK(hipFree(field->ex1))
    CHECK(hipFree(field->ey1))
    CHECK(hipFree(field->ez1))

    CHECK(hipFree(field->ex2))
    CHECK(hipFree(field->ey2))
    CHECK(hipFree(field->ez2))

    // h
    CHECK(hipFree(field->hx))
    CHECK(hipFree(field->hy))
    CHECK(hipFree(field->hz))

    // d
    CHECK(hipFree(field->dx0))
    CHECK(hipFree(field->dy0))
    CHECK(hipFree(field->dz0))

    CHECK(hipFree(field->dx1))
    CHECK(hipFree(field->dy1))
    CHECK(hipFree(field->dz1))

    CHECK(hipFree(field->dx2))
    CHECK(hipFree(field->dy2))
    CHECK(hipFree(field->dz2))

    // sigma, eps, tau
    CHECK(hipFree(field->epsI))
    CHECK(hipFree(field->epsS))
    CHECK(hipFree(field->tauD))
    CHECK(hipFree(field->sigma))

    // rp
    CHECK(hipFree(field->rpx0))
    CHECK(hipFree(field->rpy0))
    CHECK(hipFree(field->rpz0))

    CHECK(hipFree(field->rpxEnd))
    CHECK(hipFree(field->rpyEnd))
    CHECK(hipFree(field->rpzEnd))
}


void loadMaterials(FdtdParams *params, FdtdField *field, const char *specsFilePath, const char *materialsPath)
{
    // Load material specs
    int specsCount = 94;
    float *specs = (float *)calloc(specsCount * 4, sizeof(float));
    if(specs == NULL) {
        printf("Couldn't alocate %ld bytes for specs\n", (long)specsCount*4*sizeof(float));
        exit(EXIT_FAILURE);
    }
    char temp[1024];
    int index;
    float sigmaValue, epsSValue, epsIValue, tauDValue;

    FILE *specsFile = fopen(specsFilePath, "r");
    if(specsFile == NULL) {
        printf("Couldn\'t open file %s\n", specsFilePath);
        exit(EXIT_FAILURE);
    }

    for(int i=0; i<specsCount; i++) {
        fscanf(specsFile, "%d %s %g %g %g %g\n", &index, temp, &sigmaValue, &epsSValue, &epsIValue, &tauDValue);
        //printf("Read %s @ %d: %g %g %g %g\n", temp, index, sigmaValue, epsSValue, epsIValue, tauDValue);

        specs[index*4 + 0] = sigmaValue;
        specs[index*4 + 1] = epsSValue;
        specs[index*4 + 2] = epsIValue;
        specs[index*4 + 3] = tauDValue;

        if(index >= specsCount)
            break;
    }

    //fclose(specsFile);

    // Load materials
    for(int iz=0; iz<params->nz; iz++) {
        char materialFileName[1024];
        sprintf(materialFileName, "%s/v1_%05d.pgm", materialsPath, iz+1);
        FILE *materialFile = fopen(materialFileName, "r");
        
        if(materialFile == NULL) {
            printf("Couldn\'t open file %s\n", materialFileName);
            exit(EXIT_FAILURE);
        }

        //printf("Reading %s...\n", materialFileName);

        int width, height;
        fscanf(materialFile, "%s %s %s %d %d %s", temp, temp, temp, &width, &height, temp);

        for(int iy=0; iy<params->ny; iy++) {
            for(int ix=0; ix<params->nx; ix++) {
                int code;
                fscanf(materialFile, "%d", &code);

                int offset = iz*params->nx*params->ny + iy*params->nx + ix;
                field->sigma[offset] = specs[code*4 + 0];
                field->epsS[offset] =  specs[code*4 + 1];
                field->epsI[offset] =  specs[code*4 + 2];
                field->tauD[offset] =  specs[code*4 + 3];
            }
        }

        //fclose(materialFile);
    }

    //free(specs);
}


void setupMurBoundary(FdtdParams *params, FdtdField *field)
{
#ifndef __APPLE__
    int nx = params->nx;
    int ny = params->ny;
    int nz = params->nz;

    // Setup rpx
    for(int iz = 0; iz < nz; iz++) {
        for(int iy = 0; iy < ny; iy++) {
            for(int ix = 0; ix < 2; ix++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy,iz);
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) / (2.0 * params->pi * params->waveFrequency * params->eps0);

                OFFSET(field->rpx0, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +
                                                        (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }

            for(int ix = nx - 2; ix < nx; ix++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy, iz);
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) / (2.0 * params->pi * params->waveFrequency * params->eps0);
                
                OFFSET(field->rpxEnd, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +                                                  
                                                          (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }
        }
    }

    // Setup rpy
    for(int iz = 0; iz < nz; iz++) {
        for(int ix = 0; ix < nx; ix++) {
            for(int iy = 0; iy < 2; iy++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy, iz) * I;
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) /(2.0 * params->pi * params->waveFrequency * params->eps0) * I;
                
                OFFSET(field->rpy0, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +                                                      
                                                        (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }

            for(int iy = ny - 2; iy < ny; iy++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy, iz) * I;
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) / (2 * params->pi * params->waveFrequency * params->eps0) * I;
                
                OFFSET(field->rpyEnd, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +                                                      
                                                          (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }
        }
    }

    // Setup rpz
    for(int iy = 0; iy < ny; iy++) {
        for(int ix = 0; ix < nx; ix++) {
            for(int iz = 0; iz < 2; iz++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy, iz) * I;
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) / (2.0 * params->pi * params->waveFrequency * params->eps0) * I;
                
                OFFSET(field->rpz0, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +                                                  
                                                        (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }

            for(int iz = nz - 2; iz < nz; iz++) {
                float complex c1 = 0.0 + 2.0 * params->pi * params->waveFrequency * OFFSET(field->tauD, ix, iy, iz) * I;
                float complex c2 = 0.0 + OFFSET(field->sigma, ix, iy, iz) / (2.0 * params->pi * params->waveFrequency * params->eps0) * I;
                
                OFFSET(field->rpzEnd, ix, iy, iz) = creal(OFFSET(field->epsI, ix, iy, iz) +                                                  
                                                          (OFFSET(field->epsS, ix, iy, iz) - OFFSET(field->epsI, ix, iy, iz)) / (1.0 + c1) - c2);
            }
        }
    }
#endif
}


void setupSources(FdtdParams *params)
{
    int fine, temp, i2, istart;
    float *tmpdata, *tmpdata2;
    int tmpOff = 1<<16;

    params->jz = (float *)calloc(tmpOff,     sizeof(float));
    tmpdata    = (float *)calloc(tmpOff * 2, sizeof(float));
    tmpdata2   = (float *)calloc(tmpOff * 2, sizeof(float));
    
    //fine & temp
    fine = (1<<13) * params->pulseWidth * params->waveFrequency * params->dt;
    temp = 1.0/(params->pulseWidth * params->waveFrequency)/(params->dt / fine)/2.0;
    
    //tmpdata
    for(int i = -temp - 1; i <= temp + 1; i++) {
        float v1 = ((float)i/(((float)temp + 1.0)/4.0));
        float v2 = exp(-pow(v1, 2.0));
        float v3 = cos(2.0 * acos(-1.0) * params->pulseModulationFrequency * params->waveFrequency * i * (params->dt / fine));
        tmpdata[i + tmpOff] = v2 * v3;
    }

    //istart
    for(int i = -(1<<12); i < (1<<12); i++) {
         if((fabs(tmpdata[i + tmpOff]) >= 1e-9) && (i % fine == 0)) {
            istart = i;
            break;
         }
    }
    
    //setup jz 1/2
    i2 = 0;
    for(int i = istart; i <= temp+1; i += fine) {
        params->jz[i2] = tmpdata[i + tmpOff] * 1e-15 / (params->dt / 3.0);
        i2++;
    }
    
    //setup tmpdata2
    for(int i = 2; i <= (1<<14); i++) {
        float val = (((params->jz[i + 1 - 1] - params->jz[i - 1]) / params->dt) +
                    ((params->jz[i - 1] - params->jz[i - 1 - 1]) / params->dt)) / 
                    2.0 * (params->dt * params->dz) / (params->dx * params->dy * params->dz);
                                    
        tmpdata2[i - 1 + tmpOff] = val;
    }
    
    //setup jz 2/2
    for(int i=0; i < 1<<14; i++) {
        params->jz[i] = tmpdata2[i + tmpOff + 1];
    }

    free(tmpdata2);
    free(tmpdata);
}


void copyData(FdtdParams *params, FdtdField *field, FdtdField *deviceField)
{
    int n = params->nx * params->ny * params->nz * sizeof(float); 

    //e
    CHECK(hipMemcpy(deviceField->ex0, field->ex0, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->ey0, field->ey0, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->ez0, field->ez0, n, hipMemcpyHostToDevice))

    CHECK(hipMemcpy(deviceField->ex1, field->ex1, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->ey1, field->ey1, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->ez1, field->ez1, n, hipMemcpyHostToDevice))

    CHECK(hipMemcpy(deviceField->ex2, field->ex2, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->ey2, field->ey2, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->ez2, field->ez2, n, hipMemcpyHostToDevice))

    //h
    CHECK(hipMemcpy(deviceField->hx, field->hx, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->hy, field->hy, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->hz, field->hz, n, hipMemcpyHostToDevice))

    //d
    CHECK(hipMemcpy(deviceField->dx0, field->dx0, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->dy0, field->dy0, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->dz0, field->dz0, n, hipMemcpyHostToDevice))

    CHECK(hipMemcpy(deviceField->dx1, field->dx1, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->dy1, field->dy1, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->dz1, field->dz1, n, hipMemcpyHostToDevice))

    CHECK(hipMemcpy(deviceField->dx2, field->dx2, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->dy2, field->dy2, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->dz2, field->dz2, n, hipMemcpyHostToDevice))

    CHECK(hipMemcpy(deviceField->epsI, field->epsI, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->epsS, field->epsS, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->tauD, field->tauD, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->sigma, field->sigma, n, hipMemcpyHostToDevice))

    CHECK(hipMemcpy(deviceField->rpx0, field->rpx0, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->rpy0, field->rpy0, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->rpz0, field->rpz0, n, hipMemcpyHostToDevice))

    CHECK(hipMemcpy(deviceField->rpxEnd, field->rpxEnd, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->rpyEnd, field->rpyEnd, n, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(deviceField->rpzEnd, field->rpzEnd, n, hipMemcpyHostToDevice))
}


void writeResults(FdtdParams *params, FdtdField *field,
                  float *exSource, float *eySource, float *ezSource,
                  float *dxSource, float *dySource, float *dzSource,
                  int currentIteration, char *outputPath)
{
    char outputFilePath[1024];
    FILE *outputFile;

    // Used by OFFSET macro
    int nx = params->nx;
    int ny = params->ny;

    // Output x
    sprintf(outputFilePath, "%s/E_field_x_%05d.out", outputPath, currentIteration);

    outputFile = fopen(outputFilePath, "w");
    if(outputFile == NULL) {
        printf("Couldn\'t open file %s\n", outputFilePath);
        exit(EXIT_FAILURE);
    }

    for(int isrc=0; isrc < params->sourcesCount; isrc++) {
        int iy = params->sources[isrc * 3 + 1];
        int iz = params->sources[isrc * 3 + 2];
        for(int ix=0; ix < params->nx; ix++) {
            fprintf(outputFile, "%4d %4d %4d %g %g %g %g %g %g %g %g %g\n", ix, iy, iz,
                    OFFSET(dxSource, ix, iy, iz),  OFFSET(dySource, ix, iy, iz),  OFFSET(dzSource, ix, iy, iz),
                    OFFSET(field->hx, ix, iy, iz), OFFSET(field->hy, ix, iy, iz), OFFSET(field->hz, ix, iy, iz),
                    OFFSET(exSource, ix, iy, iz),  OFFSET(eySource, ix, iy, iz),  OFFSET(ezSource, ix, iy, iz));
        }
    }
    fclose(outputFile);

    // Output y
    sprintf(outputFilePath, "%s/E_field_y_%05d.out", outputPath, currentIteration);

    outputFile = fopen(outputFilePath, "w");
    if(outputFile == NULL) {
        printf("Couldn\'t open file %s\n", outputFilePath);
        exit(EXIT_FAILURE);
    }

    for(int isrc=0; isrc < params->sourcesCount; isrc++) {
        int ix = params->sources[isrc * 3 + 0];
        int iz = params->sources[isrc * 3 + 2];
        for(int iy=0; iy < params->ny; iy++) {
            fprintf(outputFile, "%4d %4d %4d %g %g %g %g %g %g %g %g %g\n", ix, iy, iz,
                    OFFSET(dxSource, ix, iy, iz),  OFFSET(dySource, ix, iy, iz),  OFFSET(dzSource, ix, iy, iz),
                    OFFSET(field->hx, ix, iy, iz), OFFSET(field->hy, ix, iy, iz), OFFSET(field->hz, ix, iy, iz),
                    OFFSET(exSource, ix, iy, iz),  OFFSET(eySource, ix, iy, iz),  OFFSET(ezSource, ix, iy, iz));
        }
    }
    fclose(outputFile);

    // Output z
    sprintf(outputFilePath, "%s/E_field_z_%05d.out", outputPath, currentIteration);

    outputFile = fopen(outputFilePath, "w");
    if(outputFile == NULL) {
        printf("Couldn\'t open file %s\n", outputFilePath);
        exit(EXIT_FAILURE);
    }

    for(int isrc=0; isrc < params->sourcesCount; isrc++) {
        int ix = params->sources[isrc * 3 + 0];
        int iy = params->sources[isrc * 3 + 1];
        for(int iz=0; iz < params->nz; iz++) {
            fprintf(outputFile, "%4d %4d %4d %g %g %g %g %g %g %g %g %g\n", ix, iy, iz,
                    OFFSET(dxSource, ix, iy, iz),  OFFSET(dySource, ix, iy, iz),  OFFSET(dzSource, ix, iy, iz),
                    OFFSET(field->hx, ix, iy, iz), OFFSET(field->hy, ix, iy, iz), OFFSET(field->hz, ix, iy, iz),
                    OFFSET(exSource, ix, iy, iz),  OFFSET(eySource, ix, iy, iz),  OFFSET(ezSource, ix, iy, iz));
        }
    }
    fclose(outputFile);
}
