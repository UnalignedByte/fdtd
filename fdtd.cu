#include "fdtd.h"

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "utils.h"


#define BLOCK_X 4
#define BLOCK_Y 4
#define BLOCK_Z 4


int main(int argc, char **argv)
{
    // Read params
    FdtdParams *params;
    params = initParamsWithPath("data/input_params");
    printParams(params);

    // Initialize field
    FdtdField  *hostField, *deviceField; // Used for CUDA

    hostField = initHostFieldWithParams(params);
    deviceField = initDeviceFieldWithParams(params);

    // Setup CUDA parameters
    dim3 gridSize = dim3((params->nx + BLOCK_X - 1)/BLOCK_X,
                         (params->ny + BLOCK_Y - 1)/BLOCK_Y,
                         (params->nz + BLOCK_Z - 1)/BLOCK_Z);
    dim3 blockSize = dim3(BLOCK_X, BLOCK_Y, BLOCK_Z);

    // Main loop
    for(int i=0; i<params->iterationsCount; i++) {
        // Run 0
        // Run 1
        // Run 2
    }

    // Clean up
    deallocDeviceField(deviceField);
    deallocHostField(hostField);
    deallocParams(params);
}


void printUsage()
{
}


FdtdParams *initParamsWithPath(const char *filePath)
{
    FdtdParams *params = (FdtdParams *)malloc(sizeof(FdtdParams));
    params->inputPath = (char *)malloc(sizeof(char) * 1024);
    params->outputPath = (char *)malloc(sizeof(char) * 1024);

    FILE *paramsFile = fopen(filePath, "r");
    //check(paramsFile != NULL, "Cannot open file");
    
    int tempLength = 1024;
    char temp[tempLength];

    //nx ny nz (field size)
    fscanf(paramsFile, "%s %d %d %d\n", temp, &params->nx, &params->ny, &params->nz);
    //t_max (simulation runs count)
    fscanf(paramsFile, "%s %d\n", temp, &params->iterationsCount);
    //unused (nf)
    fgets(temp, tempLength, paramsFile);
    //env_set_dir (input path)
    fscanf(paramsFile, "%s %s\n", temp, params->inputPath);
    //unused (env_file_prefix)
    fgets(temp, tempLength, paramsFile);
    //output_dir (output path) 
    fscanf(paramsFile, "%s %s\n", temp, params->outputPath);
    //unused (output_format)
    fgets(temp, tempLength, paramsFile);
    //unused (impulse_resp_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (pec_flag) 
    fgets(temp, tempLength, paramsFile);
    //unused (read_env_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (output_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (bzip2_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (output_start)
    fgets(temp, tempLength, paramsFile);
    //unused (output_finish)
    fgets(temp, tempLength, paramsFile);
    //unused (source_type)
    fgets(temp, tempLength, paramsFile);
    //elements_per_wavelength
    fscanf(paramsFile, "%s %d\n", temp, &params->elementsPerWave);
    //wave_freq
    fscanf(paramsFile, "%s %g\n", temp, &params->waveFrequency);
    //pulse_width
    fscanf(paramsFile, "%s %g\n", temp, &params->pulseWidth);
    //pulse_modulation_frequency
    fscanf(paramsFile, "%s %g\n", temp, &params->pulseModulationFrequency);
    //number_of_excitation_sources
    fscanf(paramsFile, "%s %d\n", temp, &params->sourcesCount);
    //source_location
    params->sources = (int *)malloc(sizeof(int) * params->sourcesCount * 3);
    for(int i=0; i<params->sourcesCount; i++) {
        fscanf(paramsFile, "%s %d %d %d\n", temp,
                                            &params->sources[i*3],
                                            &params->sources[i*3 + 1],
                                            &params->sources[i*3 + 2]);
    }
    //unused (pulse_type)
    fgets(temp, tempLength, paramsFile);
    //fsigma (sigma)
    fscanf(paramsFile, "%s %f\n", temp, &params->sigma);
    //feps_s (eps_s)
    fscanf(paramsFile, "%s %f\n", temp, &params->eps_s);
    //feps_inf (eps_i)
    fscanf(paramsFile, "%s %f\n", temp, &params->eps_i);
    //ftau_d (tau_d)
    fscanf(paramsFile, "%s %f\n", temp, &params->tau_d);
    
    fclose(paramsFile);

    return params;
}


void deallocParams(FdtdParams *params)
{
    free(params->inputPath);
    free(params->outputPath);
    free(params);
}


void printParams(FdtdParams *params)
{
    printf("Field size:                 %dx%dx%d\n", params->nx, params->ny, params->nz);
    printf("Iterations count:           %d\n", params->iterationsCount);
    printf("Input path:                 %s\n", params->inputPath);
    printf("Output path:                %s\n", params->outputPath);
    printf("Elements per wavelength:    %d\n", params->elementsPerWave);
    printf("Wave frequency:             %g\n", params->waveFrequency);
    printf("Pulse width:                %g\n", params->pulseWidth);
    printf("Pulse modulation frequency: %g\n", params->pulseModulationFrequency);
    printf("Sources count:              %d\n", params->sourcesCount);
    for(int i=0; i<params->sourcesCount; i++)
        printf("Source position:            %dx%dx%d\n", params->sources[i*3],
                                                         params->sources[i*3 + 1],
                                                         params->sources[i*3 + 2]);
    printf("Default sigma:              %g\n", params->sigma);
    printf("Default eps_s:              %g\n", params->eps_s);
    printf("Default eps_i:              %g\n", params->eps_i);
    printf("Default tau_d:              %g\n", params->tau_d);
    printf("\n");
}


FdtdField  *initHostFieldWithParams(FdtdParams *params)
{
    int n = params->nx * params->ny * params->nz * sizeof(float); 

    FdtdField *field = (FdtdField *)malloc(sizeof(FdtdField));

    CHECK(hipHostAlloc(&field->ex1, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ey1, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ez1, n, hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->ex2, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ey2, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ez2, n, hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->ex3, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ey3, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ez3, n, hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->hx, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->hy, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->hz, n, hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->dx1, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dy1, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dz1, n, hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->dx2, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dy2, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dz2, n, hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->dx3, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dy3, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dz3, n, hipHostMallocDefault))

    return field;
}


void deallocHostField(FdtdField *field)
{
    CHECK(hipFree(field->ex1))
    CHECK(hipFree(field->ey1))
    CHECK(hipFree(field->ez1))

    CHECK(hipFree(field->ex2))
    CHECK(hipFree(field->ey2))
    CHECK(hipFree(field->ez2))

    CHECK(hipFree(field->ex3))
    CHECK(hipFree(field->ey3))
    CHECK(hipFree(field->ez3))

    CHECK(hipFree(field->hx))
    CHECK(hipFree(field->hy))
    CHECK(hipFree(field->hz))

    CHECK(hipFree(field->dx1))
    CHECK(hipFree(field->dy1))
    CHECK(hipFree(field->dz1))

    CHECK(hipFree(field->dx2))
    CHECK(hipFree(field->dy2))
    CHECK(hipFree(field->dz2))

    CHECK(hipFree(field->dx3))
    CHECK(hipFree(field->dy3))
    CHECK(hipFree(field->dz3))

    free(field);
}


FdtdField *initDeviceFieldWithParams(FdtdParams *params)
{
    int n = params->nx * params->ny * params->nz * sizeof(float); 

    FdtdField *field = (FdtdField *)malloc(sizeof(FdtdField));

    CHECK(hipMalloc(&field->ex1, n))
    CHECK(hipMalloc(&field->ey1, n))
    CHECK(hipMalloc(&field->ez1, n))

    CHECK(hipMalloc(&field->ex2, n))
    CHECK(hipMalloc(&field->ey2, n))
    CHECK(hipMalloc(&field->ez2, n))

    CHECK(hipMalloc(&field->ex3, n))
    CHECK(hipMalloc(&field->ey3, n))
    CHECK(hipMalloc(&field->ez3, n))

    CHECK(hipMalloc(&field->hx, n))
    CHECK(hipMalloc(&field->hy, n))
    CHECK(hipMalloc(&field->hz, n))

    CHECK(hipMalloc(&field->dx1, n))
    CHECK(hipMalloc(&field->dy1, n))
    CHECK(hipMalloc(&field->dz1, n))

    CHECK(hipMalloc(&field->dx2, n))
    CHECK(hipMalloc(&field->dy2, n))
    CHECK(hipMalloc(&field->dz2, n))

    CHECK(hipMalloc(&field->dx3, n))
    CHECK(hipMalloc(&field->dy3, n))
    CHECK(hipMalloc(&field->dz3, n))

    CHECK(hipMalloc(&field->eps_i, n))
    CHECK(hipMalloc(&field->eps_s, n))
    CHECK(hipMalloc(&field->tau_d, n))
    CHECK(hipMalloc(&field->sigma, n))

    CHECK(hipMalloc(&field->rp_x_0, n))
    CHECK(hipMalloc(&field->rp_y_0, n))
    CHECK(hipMalloc(&field->rp_z_0, n))

    CHECK(hipMalloc(&field->rp_x_end, n))
    CHECK(hipMalloc(&field->rp_y_end, n))
    CHECK(hipMalloc(&field->rp_z_end, n))

    return field;
}


void deallocDeviceField(FdtdField *field)
{
    CHECK(hipFree(field->ex1))
    CHECK(hipFree(field->ey1))
    CHECK(hipFree(field->ez1))

    CHECK(hipFree(field->ex2))
    CHECK(hipFree(field->ey2))
    CHECK(hipFree(field->ez2))

    CHECK(hipFree(field->ex3))
    CHECK(hipFree(field->ey3))
    CHECK(hipFree(field->ez3))

    CHECK(hipFree(field->hx))
    CHECK(hipFree(field->hy))
    CHECK(hipFree(field->hz))

    CHECK(hipFree(field->dx1))
    CHECK(hipFree(field->dy1))
    CHECK(hipFree(field->dz1))

    CHECK(hipFree(field->dx2))
    CHECK(hipFree(field->dy2))
    CHECK(hipFree(field->dz2))

    CHECK(hipFree(field->dx3))
    CHECK(hipFree(field->dy3))
    CHECK(hipFree(field->dz3))

    CHECK(hipFree(field->eps_i))
    CHECK(hipFree(field->eps_s))
    CHECK(hipFree(field->tau_d))
    CHECK(hipFree(field->sigma))

    CHECK(hipFree(field->rp_x_0))
    CHECK(hipFree(field->rp_y_0))
    CHECK(hipFree(field->rp_z_0))

    CHECK(hipFree(field->rp_x_end))
    CHECK(hipFree(field->rp_y_end))
    CHECK(hipFree(field->rp_z_end))
}
