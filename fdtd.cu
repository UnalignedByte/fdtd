#include "fdtd.h"

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "utils.h"


#define BLOCK_X 4
#define BLOCK_Y 4
#define BLOCK_Z 4


int main(int argc, char **argv)
{
    // Read params
    FdtdParams *params;
    printf("Reading parameters...\n");
    params = initParamsWithPath("data/input_params");
    printParams(params);

    // Initialize field
    FdtdField  *field, *deviceField; // Used for CUDA

    printf("Initializing field...\n");
    field = initFieldWithParams(params);
    deviceField = initDeviceFieldWithParams(params);

    printf("Reading materials data...\n");
    loadMaterials(params, field, "data/mat_specs_riken", params->inputPath);
    printf("Setting up mur boundary...\n");
    setupMurBoundary(params, field);

    // Setup CUDA parameters
    dim3 gridSize = dim3((params->nx + BLOCK_X - 1)/BLOCK_X,
                         (params->ny + BLOCK_Y - 1)/BLOCK_Y,
                         (params->nz + BLOCK_Z - 1)/BLOCK_Z);
    dim3 blockSize = dim3(BLOCK_X, BLOCK_Y, BLOCK_Z);

    // Main loop
    for(int i=0; i<params->iterationsCount; i++) {
        // Run 0
        printf("Running iteration %d", i);

        // Run 1
        printf("Running iteration %d", i+1);

        // Run 2
        printf("Running iteration %d", i+2);
    }

    // Clean up
    deallocDeviceField(deviceField);
    deallocField(field);
    deallocParams(params);
}


FdtdParams *initParamsWithPath(const char *filePath)
{
    FdtdParams *params = (FdtdParams *)malloc(sizeof(FdtdParams));
    params->inputPath = (char *)malloc(sizeof(char) * 1024);
    params->outputPath = (char *)malloc(sizeof(char) * 1024);

    FILE *paramsFile = fopen(filePath, "r");
    //check(paramsFile != NULL, "Cannot open file");
    
    int tempLength = 1024;
    char temp[tempLength];

    //nx ny nz (field size)
    fscanf(paramsFile, "%s %d %d %d\n", temp, &params->nx, &params->ny, &params->nz);
    //t_max (simulation runs count)
    fscanf(paramsFile, "%s %d\n", temp, &params->iterationsCount);
    //unused (nf)
    fgets(temp, tempLength, paramsFile);
    //env_set_dir (input path)
    fscanf(paramsFile, "%s %s\n", temp, params->inputPath);
    //unused (env_file_prefix)
    fgets(temp, tempLength, paramsFile);
    //output_dir (output path) 
    fscanf(paramsFile, "%s %s\n", temp, params->outputPath);
    //unused (output_format)
    fgets(temp, tempLength, paramsFile);
    //unused (impulse_resp_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (pec_flag) 
    fgets(temp, tempLength, paramsFile);
    //unused (read_env_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (output_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (bzip2_flag)
    fgets(temp, tempLength, paramsFile);
    //unused (output_start)
    fgets(temp, tempLength, paramsFile);
    //unused (output_finish)
    fgets(temp, tempLength, paramsFile);
    //unused (source_type)
    fgets(temp, tempLength, paramsFile);
    //elements_per_wavelength
    fscanf(paramsFile, "%s %d\n", temp, &params->elementsPerWave);
    //wave_freq
    fscanf(paramsFile, "%s %g\n", temp, &params->waveFrequency);
    //pulse_width
    fscanf(paramsFile, "%s %g\n", temp, &params->pulseWidth);
    //pulse_modulation_frequency
    fscanf(paramsFile, "%s %g\n", temp, &params->pulseModulationFrequency);
    //number_of_excitation_sources
    fscanf(paramsFile, "%s %d\n", temp, &params->sourcesCount);
    //source_location
    params->sources = (int *)malloc(sizeof(int) * params->sourcesCount * 3);
    for(int i=0; i<params->sourcesCount; i++) {
        fscanf(paramsFile, "%s %d %d %d\n", temp,
                                            &params->sources[i*3],
                                            &params->sources[i*3 + 1],
                                            &params->sources[i*3 + 2]);
    }
    //unused (pulse_type)
    fgets(temp, tempLength, paramsFile);
    //fsigma (sigma)
    fscanf(paramsFile, "%s %f\n", temp, &params->sigma);
    //feps_s (eps_s)
    fscanf(paramsFile, "%s %f\n", temp, &params->eps_s);
    //feps_inf (eps_i)
    fscanf(paramsFile, "%s %f\n", temp, &params->eps_i);
    //ftau_d (tau_d)
    fscanf(paramsFile, "%s %f\n", temp, &params->tau_d);
    
    fclose(paramsFile);

    return params;
}


void deallocParams(FdtdParams *params)
{
    free(params->inputPath);
    free(params->outputPath);
    free(params);
}


void printParams(FdtdParams *params)
{
    printf("Field size:                 %dx%dx%d\n", params->nx, params->ny, params->nz);
    printf("Iterations count:           %d\n", params->iterationsCount);
    printf("Input path:                 %s\n", params->inputPath);
    printf("Output path:                %s\n", params->outputPath);
    printf("Elements per wavelength:    %d\n", params->elementsPerWave);
    printf("Wave frequency:             %g\n", params->waveFrequency);
    printf("Pulse width:                %g\n", params->pulseWidth);
    printf("Pulse modulation frequency: %g\n", params->pulseModulationFrequency);
    printf("Sources count:              %d\n", params->sourcesCount);
    for(int i=0; i<params->sourcesCount; i++)
        printf("Source position:            %dx%dx%d\n", params->sources[i*3],
                                                         params->sources[i*3 + 1],
                                                         params->sources[i*3 + 2]);
    printf("Default sigma:              %g\n", params->sigma);
    printf("Default eps_s:              %g\n", params->eps_s);
    printf("Default eps_i:              %g\n", params->eps_i);
    printf("Default tau_d:              %g\n", params->tau_d);
    printf("\n");
}


FdtdField *initFieldWithParams(FdtdParams *params)
{
    int n = params->nx * params->ny * params->nz * sizeof(float); 

    FdtdField *field = (FdtdField *)malloc(sizeof(FdtdField));

    // e
    CHECK(hipHostAlloc(&field->ex1, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ey1, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ez1, n, hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->ex2, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ey2, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ez2, n, hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->ex3, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ey3, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->ez3, n, hipHostMallocDefault))

    // h
    CHECK(hipHostAlloc(&field->hx, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->hy, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->hz, n, hipHostMallocDefault))

    // d
    CHECK(hipHostAlloc(&field->dx1, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dy1, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dz1, n, hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->dx2, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dy2, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dz2, n, hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->dx3, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dy3, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->dz3, n, hipHostMallocDefault))

    // sigma, eps, tau
    CHECK(hipHostAlloc(&field->sigma, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->eps_s, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->eps_i, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->tau_d, n, hipHostMallocDefault))

    // rp
    CHECK(hipHostAlloc(&field->rp_x_0, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->rp_y_0, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->rp_z_0, n, hipHostMallocDefault))

    CHECK(hipHostAlloc(&field->rp_x_end, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->rp_y_end, n, hipHostMallocDefault))
    CHECK(hipHostAlloc(&field->rp_z_end, n, hipHostMallocDefault))

    return field;
}


void deallocField(FdtdField *field)
{
    // e
    CHECK(hipFree(field->ex1))
    CHECK(hipFree(field->ey1))
    CHECK(hipFree(field->ez1))

    CHECK(hipFree(field->ex2))
    CHECK(hipFree(field->ey2))
    CHECK(hipFree(field->ez2))

    CHECK(hipFree(field->ex3))
    CHECK(hipFree(field->ey3))
    CHECK(hipFree(field->ez3))

    // h
    CHECK(hipFree(field->hx))
    CHECK(hipFree(field->hy))
    CHECK(hipFree(field->hz))

    // d
    CHECK(hipFree(field->dx1))
    CHECK(hipFree(field->dy1))
    CHECK(hipFree(field->dz1))

    CHECK(hipFree(field->dx2))
    CHECK(hipFree(field->dy2))
    CHECK(hipFree(field->dz2))

    CHECK(hipFree(field->dx3))
    CHECK(hipFree(field->dy3))
    CHECK(hipFree(field->dz3))

    // sigma, eps, tau
    CHECK(hipFree(&field->sigma))
    CHECK(hipFree(&field->eps_s))
    CHECK(hipFree(&field->eps_i))
    CHECK(hipFree(&field->tau_d))

    // rp
    CHECK(hipFree(&field->rp_x_0))
    CHECK(hipFree(&field->rp_y_0))
    CHECK(hipFree(&field->rp_z_0))

    CHECK(hipFree(&field->rp_x_end))
    CHECK(hipFree(&field->rp_y_end))
    CHECK(hipFree(&field->rp_z_end))

    free(field);
}


FdtdField *initDeviceFieldWithParams(FdtdParams *params)
{
    int n = params->nx * params->ny * params->nz * sizeof(float); 

    FdtdField *field = (FdtdField *)malloc(sizeof(FdtdField));

    // e
    CHECK(hipMalloc(&field->ex1, n))
    CHECK(hipMalloc(&field->ey1, n))
    CHECK(hipMalloc(&field->ez1, n))

    CHECK(hipMalloc(&field->ex2, n))
    CHECK(hipMalloc(&field->ey2, n))
    CHECK(hipMalloc(&field->ez2, n))

    CHECK(hipMalloc(&field->ex3, n))
    CHECK(hipMalloc(&field->ey3, n))
    CHECK(hipMalloc(&field->ez3, n))

    // h
    CHECK(hipMalloc(&field->hx, n))
    CHECK(hipMalloc(&field->hy, n))
    CHECK(hipMalloc(&field->hz, n))

    // d
    CHECK(hipMalloc(&field->dx1, n))
    CHECK(hipMalloc(&field->dy1, n))
    CHECK(hipMalloc(&field->dz1, n))

    CHECK(hipMalloc(&field->dx2, n))
    CHECK(hipMalloc(&field->dy2, n))
    CHECK(hipMalloc(&field->dz2, n))

    CHECK(hipMalloc(&field->dx3, n))
    CHECK(hipMalloc(&field->dy3, n))
    CHECK(hipMalloc(&field->dz3, n))

    // sigma, eps, tau
    CHECK(hipMalloc(&field->eps_i, n))
    CHECK(hipMalloc(&field->eps_s, n))
    CHECK(hipMalloc(&field->tau_d, n))
    CHECK(hipMalloc(&field->sigma, n))

    // rp
    CHECK(hipMalloc(&field->rp_x_0, n))
    CHECK(hipMalloc(&field->rp_y_0, n))
    CHECK(hipMalloc(&field->rp_z_0, n))

    CHECK(hipMalloc(&field->rp_x_end, n))
    CHECK(hipMalloc(&field->rp_y_end, n))
    CHECK(hipMalloc(&field->rp_z_end, n))

    return field;
}


void deallocDeviceField(FdtdField *field)
{
    // e
    CHECK(hipFree(field->ex1))
    CHECK(hipFree(field->ey1))
    CHECK(hipFree(field->ez1))

    CHECK(hipFree(field->ex2))
    CHECK(hipFree(field->ey2))
    CHECK(hipFree(field->ez2))

    CHECK(hipFree(field->ex3))
    CHECK(hipFree(field->ey3))
    CHECK(hipFree(field->ez3))

    // h
    CHECK(hipFree(field->hx))
    CHECK(hipFree(field->hy))
    CHECK(hipFree(field->hz))

    // d
    CHECK(hipFree(field->dx1))
    CHECK(hipFree(field->dy1))
    CHECK(hipFree(field->dz1))

    CHECK(hipFree(field->dx2))
    CHECK(hipFree(field->dy2))
    CHECK(hipFree(field->dz2))

    CHECK(hipFree(field->dx3))
    CHECK(hipFree(field->dy3))
    CHECK(hipFree(field->dz3))

    // sigma, eps, tau
    CHECK(hipFree(field->eps_i))
    CHECK(hipFree(field->eps_s))
    CHECK(hipFree(field->tau_d))
    CHECK(hipFree(field->sigma))

    // rp
    CHECK(hipFree(field->rp_x_0))
    CHECK(hipFree(field->rp_y_0))
    CHECK(hipFree(field->rp_z_0))

    CHECK(hipFree(field->rp_x_end))
    CHECK(hipFree(field->rp_y_end))
    CHECK(hipFree(field->rp_z_end))
}


void loadMaterials(FdtdParams *params, FdtdField *field, const char *specsFilePath, const char *materialsPath)
{
    // Load material specs
    int specsCount = 94;
    float *specs = (float *)calloc(specsCount * 4, sizeof(float));
    char temp[1024];
    int index;
    float sigma_value, eps_s_value, eps_i_value, tau_d_value;

    FILE *specsFile = fopen(specsFilePath, "r");
    //check(specsFile != NULL, "Cannot open specs file");

    for(int i=0; i<94; i++) {
        fscanf(specsFile, "%d %s %g %g %g %g", &index, temp, &sigma_value, &eps_s_value, &eps_i_value, &tau_d_value);
        specs[index*4 + 0] = sigma_value;
        specs[index*4 + 1] = eps_s_value;
        specs[index*4 + 2] = eps_i_value;
        specs[index*4 + 3] = tau_d_value;
    }

    fclose(specsFile);

    // Load materials
    for(int iz=0; iz<params->nz; iz++) {
        char materialFileName[1024];
        sprintf(materialFileName, "%s/v1_%5d.pgm", materialsPath, iz);
        FILE *materialFile = fopen(materialFileName, "r");

        printf("Reading %s...", materialFileName);

        int width, height;
        fscanf(materialFile, "%s %s %s %d %d %s", temp, temp, temp, &width, &height, temp);

        for(int iy=0; iy<params->ny; iy++) {
            for(int ix=0; ix<params->nx; ix++) {
                int code;
                fscanf(materialFile, "%d", &code);

                int offset = iz*params->nx*params->ny + iy*params->nx + ix;
                field->sigma[offset] = specs[code*4 + 0];
                field->eps_s[offset] = specs[code*4 + 1];
                field->eps_i[offset] = specs[code*4 + 2];
                field->tau_d[offset] = specs[code*4 + 3];
            }
        }

        fclose(materialFile);
    }

    free(specs);
}


void setupMurBoundary(FdtdParams *params, FdtdField *field)
{
    // Setup rp_x
    for(int iz = 0; iz < params->nz; iz++) {
        for(int iy = 0; iy < params->ny; iy++) {
            for(int ix = 0; ix < 2; ix++) {
                int offset = iz * params->nx * params->ny + iy * params->nx + ix;

                field->rp_x_0[offset] = 0.0; 
            }

            for(int ix = params->nx - 2; ix < params->nx; ix++) {
                int offset = iz * params->nx * params->ny + iy * params->nx + ix;

                field->rp_x_end[offset] = 0.0;
            }
        }
    }

    // Setup rp_y
    for(int iz = 0; iz < params->nz; iz++) {
        for(int ix = 0; ix < params->nx; ix++) {
            for(int iy = 0; iy < 2; iy++) {
                int offset = iz * params->nx * params->ny + iy * params->nx + ix;

                field->rp_y_0[offset] = 0.0; 
            }

            for(int iy = params->ny - 2; iy < params->ny; iy++) {
                int offset = iz * params->nx * params->ny + iy * params->nx + ix;

                field->rp_y_end[offset] = 0.0;
            }
        }
    }

    // Setup rp_z
    for(int iy = 0; iy < params->ny; iy++) {
        for(int ix = 0; ix < params->nx; ix++) {
            for(int iz = 0; iz < 2; iz++) {
                int offset = iz * params->nx * params->ny + iy * params->nx + ix;

                field->rp_z_0[offset] = 0.0; 
            }

            for(int iz = params->nz - 2; iz < params->nz; iz++) {
                int offset = iz * params->nx * params->ny + iy * params->nx + ix;

                field->rp_z_end[offset] = 0.0;
            }
        }
    }
}
